#include "hip/hip_runtime.h"
//#define CHECKRC

#include <iostream>
#include <vector>

#include "connect.h"
#include "remote_connect.h"
#include "utilities.h"

// INITIALIZATION
//
// Define two arrays that map remote source nodes to local spike buffers
// There is one element for each remote host,
// so the array size is n_hosts
// Each of the two arrays contain n_remote_source_node_map elements
// that represent a map, with n_remote_source_node_map pairs
// (remote node index, local spike buffer index)
// where n_remote_source_node_map is the number of nodes in the source host
// that have outgoing connections to local nodes.
// All elements are initially empty:
// n_remote_source_nodes[i_source_host] = 0 for each i_source_host
// The map is organized in blocks each with node_map_block_size
// elements, which are allocated dynamically

__constant__ uint node_map_block_size; // = 100000;
uint h_node_map_block_size; // = 100000;

// number of elements in the map for each source host
// n_remote_source_node_map[i_source_host]
// with i_source_host = 0, ..., n_hosts-1 excluding this host itself
__device__ uint *n_remote_source_node_map; // [n_hosts];
uint *d_n_remote_source_node_map;
std::vector<uint> h_n_remote_source_node_map;

// remote_source_node_map[i_source_host][i_block][i]
std::vector< std::vector<uint*> > h_remote_source_node_map;
__device__ uint ***remote_source_node_map;

// local_spike_buffer_map[i_source_host][i_block][i]
std::vector< std::vector<uint*> > h_local_spike_buffer_map;
__device__ uint ***local_spike_buffer_map;
uint ***d_local_spike_buffer_map;
// hd_local_spike_buffer_map[i_source_host] vector of pointers to gpu memory
std::vector<uint**> hd_local_spike_buffer_map;

// Define two arrays that map local source nodes to remote spike buffers.
// The structure is the same as for remote source nodes

// number of elements in the map for each target host
// n_local_source_node_map[i_target_host]
// with i_target_host = 0, ..., n_hosts-1 excluding this host itself
__device__ uint *n_local_source_node_map; // [n_hosts]; 
uint *d_n_local_source_node_map;
std::vector<uint> h_n_local_source_node_map;

// local_source_node_map[i_target_host][i_block][i]
std::vector< std::vector<uint*> > h_local_source_node_map;
__device__ uint ***local_source_node_map;
uint ***d_local_source_node_map;
// hd_local_source_node_map[i_target_host] vector of pointers to gpu memory
std::vector<uint**> hd_local_source_node_map;


// number of remote target hosts on which each local node
// has outgoing connections. Must be initially set to 0
uint *d_n_target_hosts; // [n_nodes] 
// cumulative sum of d_n_target_hosts
uint *d_n_target_hosts_cumul; // [n_nodes+1]

// Global array with remote target hosts indexes of all nodes
// target_host_array[total_num] where total_num is the sum
// of n_target_hosts[i_node] on all nodes
uint *d_target_host_array;
// pointer to the starting position in target_host_array
// of the target hosts for the node i_node
uint **d_node_target_hosts; // [i_node]

// Global array with remote target hosts map indexes of all nodes
// target_host_i_map[total_num] where total_num is the sum
// of n_target_hosts[i_node] on all nodes
uint *d_target_host_i_map;
// pointer to the starting position in target_host_i_map array
// of the target host map indexes for the node i_node
uint **d_node_target_host_i_map; // [i_node]

// node map index
uint **d_node_map_index; // [i_node]

// Define a boolean array with one boolean value for each connection rule
// - true if the rule always creates at least one outgoing connection
// from each source node (one_to_one, all_to_all, fixed_outdegree)
// - false otherwise (fixed_indegree, fixed_total_number, pairwise_bernoulli)
bool *use_all_source_nodes; // [n_connection_rules]:

__constant__ uint n_local_nodes; // number of local nodes

// Allocate GPU memory for new remote-source-node-map blocks
int allocRemoteSourceNodeMapBlocks(std::vector<uint*> &i_remote_src_node_map,
				   std::vector<uint*> &i_local_spike_buf_map,
				   int64_t block_size, uint new_n_block)
{
  // allocate new blocks if needed
  for (uint ib=i_remote_src_node_map.size(); ib<new_n_block; ib++) {
    uint *d_remote_src_node_blk_pt;
    uint *d_local_spike_buf_blk_pt;
    // allocate GPU memory for new blocks 
    CUDAMALLOCCTRL("&d_remote_src_node_blk_pt",&d_remote_src_node_blk_pt, block_size*sizeof(uint));
    CUDAMALLOCCTRL("&d_local_spike_buf_blk_pt",&d_local_spike_buf_blk_pt, block_size*sizeof(uint));
      
    i_remote_src_node_map.push_back(d_remote_src_node_blk_pt);
    i_local_spike_buf_map.push_back(d_local_spike_buf_blk_pt);
  }
    
  return 0;
}

// Allocate GPU memory for new local-source-node-map blocks
int allocLocalSourceNodeMapBlocks(std::vector<uint*> &i_local_src_node_map,
				  int64_t block_size, uint new_n_block)
{
  // allocate new blocks if needed
  for (uint ib=i_local_src_node_map.size(); ib<new_n_block; ib++) {
    uint *d_local_src_node_blk_pt;
    // allocate GPU memory for new blocks 
    CUDAMALLOCCTRL("&d_local_src_node_blk_pt",&d_local_src_node_blk_pt, block_size*sizeof(uint));
      
    i_local_src_node_map.push_back(d_local_src_node_blk_pt);
  }
    
  return 0;
}


// Initialize the maps for n_hosts hosts
int RemoteConnectionMapInit(int n_hosts)
{
#ifdef CHECKRC
  h_node_map_block_size = 3; // initialize node map block size
#else
  h_node_map_block_size = 10000; // initialize node map block size
#endif

  hipMemcpyToSymbol(HIP_SYMBOL(node_map_block_size), &h_node_map_block_size, sizeof(uint));

  // allocate and init to 0 n. of elements in the map for each source host
  CUDAMALLOCCTRL("&d_n_remote_source_node_map",&d_n_remote_source_node_map, n_hosts*sizeof(uint));
  gpuErrchk(hipMemset(d_n_remote_source_node_map, 0, n_hosts*sizeof(uint)));

  // allocate and init to 0 n. of elements in the map for each source host
  CUDAMALLOCCTRL("&d_n_local_source_node_map",&d_n_local_source_node_map, n_hosts*sizeof(uint));
  gpuErrchk(hipMemset(d_n_local_source_node_map, 0, n_hosts*sizeof(uint)));

  // initialize maps
  for (int i_host=0; i_host<n_hosts; i_host++) {
    std::vector<uint*> rsn_map;
    h_remote_source_node_map.push_back(rsn_map);
      
    std::vector<uint*> lsb_map;
    h_local_spike_buffer_map.push_back(lsb_map);

    std::vector<uint*> lsn_map;
    h_local_source_node_map.push_back(lsn_map);
  }
    

  // launch kernel to copy pointers to CUDA variables ?? maybe in calibration?
  // .....
  //RemoteConnectionMapInitKernel // <<< , >>>
  //  (d_n_remote_source_node_map,
  //   d_remote_source_node_map,
  //   d_local_spike_buffer_map,
  //   d_n_local_source_node_map,
  //   d_local_source_node_map);
    
  return 0;
}


__global__ void setTargetHostArrayNodePointersKernel
(uint *target_host_array, uint *target_host_i_map, uint *n_target_hosts_cumul,
 uint **node_target_hosts, uint **node_target_host_i_map, uint n_nodes)
{
  uint i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node>=n_nodes) return;
  node_target_hosts[i_node] = target_host_array + n_target_hosts_cumul[i_node];
  node_target_host_i_map[i_node] = target_host_i_map
    + n_target_hosts_cumul[i_node];
}


// kernel that fills the arrays target_host_array
// and target_host_i_map using the node map
__global__ void fillTargetHostArrayFromMapKernel
(uint **node_map, uint n_node_map, uint *count_mapped, uint **node_target_hosts,
 uint **node_target_host_i_map, uint n_nodes, uint i_target_host)
{
  uint i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node>=n_nodes) return;
  uint i_block;
  uint i_in_block;
  // check if node index is in map
  bool mapped = checkIfValueIsIn2DArr(i_node, node_map,
				      n_node_map, node_map_block_size,
				      &i_block, &i_in_block);
  // If it is mapped
  if (mapped) {
    uint i_node_map = i_block*node_map_block_size + i_in_block;
    uint pos = count_mapped[i_node]++;
    node_target_host_i_map[i_node][pos] = i_node_map;
    node_target_hosts[i_node][pos] = i_target_host;  
  }
}


// Calibrate the maps
int  NESTGPU::RemoteConnectionMapCalibrate(int i_host, int n_hosts)
{
  //std::cout << "In RemoteConnectionMapCalibrate " << i_host << " "
  //	    << n_hosts << "\n";
  // vector of pointers to local source node maps in device memory
  // per target host hd_local_source_node_map[target_host]
  // type std::vector<uint*>
  // set its size and initialize to NULL
  hd_local_source_node_map.resize(n_hosts, NULL);
  // number of elements in each local source node map
  // h_n_local_source_node_map[target_host]
  // set its size and initialize to 0
  h_n_local_source_node_map.resize(n_hosts, 0);
  // vector of pointers to local spike buffer maps in device memory
  // per source host hd_local_spike_buffer_map[source_host]
  // type std::vector<int*>
  // set its size and initialize to NULL
  hd_local_spike_buffer_map.resize(n_hosts, NULL);
  // number of elements in each remote-source-node->local-spike-buffer map
  // h_n_remote_source_node_map[source_host]
  // set its size and initialize to 0
  h_n_remote_source_node_map.resize(n_hosts, 0);
  // loop on target hosts, skip self host
  for (int tg_host=0; tg_host<n_hosts; tg_host++) {
    if (tg_host != i_host) {
      // get number of elements in each map from device memory
      uint n_node_map;
      gpuErrchk(hipMemcpy(&n_node_map,
			   &d_n_local_source_node_map[tg_host], sizeof(uint),
			   hipMemcpyDeviceToHost));
      // put it in h_n_local_source_node_map[tg_host]
      h_n_local_source_node_map[tg_host] = n_node_map;
      // Allocate array of local source node map blocks
      // and copy their address from host to device
      hd_local_source_node_map[tg_host] = NULL;
      uint n_blocks = h_local_source_node_map[tg_host].size();
      if (n_blocks>0) {
	CUDAMALLOCCTRL("&hd_local_source_node_map[tg_host]",&hd_local_source_node_map[tg_host],
			     n_blocks*sizeof(uint*));
	gpuErrchk(hipMemcpy(hd_local_source_node_map[tg_host],
			     &h_local_source_node_map[tg_host][0],
			     n_blocks*sizeof(uint*),
			     hipMemcpyHostToDevice));
      }
    }
  }
  // allocate d_local_source_node_map and copy it from host to device
  CUDAMALLOCCTRL("&d_local_source_node_map",&d_local_source_node_map, n_hosts*sizeof(uint**));
  gpuErrchk(hipMemcpy(d_local_source_node_map, &hd_local_source_node_map[0],
		       n_hosts*sizeof(uint**), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(local_source_node_map),
			       &d_local_source_node_map, sizeof(uint***)));

  // loop on source hosts, skip self host
  for (int src_host=0; src_host<n_hosts; src_host++) {
    if (src_host != i_host) {
      // get number of elements in each map from device memory
      uint n_node_map;
      gpuErrchk(hipMemcpy(&n_node_map,
			   &d_n_remote_source_node_map[src_host], sizeof(uint),
			   hipMemcpyDeviceToHost));
      // put it in h_n_remote_source_node_map[src_host]
      h_n_remote_source_node_map[src_host] = n_node_map;
      // Allocate array of local spike buffer map blocks
      // and copy their address from host to device
      uint n_blocks = h_local_spike_buffer_map[src_host].size();
      hd_local_spike_buffer_map[src_host] = NULL;
      if (n_blocks>0) {
	CUDAMALLOCCTRL("&hd_local_spike_buffer_map[src_host]",&hd_local_spike_buffer_map[src_host],
			     n_blocks*sizeof(uint*));
	gpuErrchk(hipMemcpy(hd_local_spike_buffer_map[src_host],
			     &h_local_spike_buffer_map[src_host][0],
			     n_blocks*sizeof(uint*),
			     hipMemcpyHostToDevice));
      }
    }
  }
  // allocate d_local_spike_buffer_map and copy it from host to device
  CUDAMALLOCCTRL("&d_local_spike_buffer_map",&d_local_spike_buffer_map, n_hosts*sizeof(uint**));
  gpuErrchk(hipMemcpy(d_local_spike_buffer_map, &hd_local_spike_buffer_map[0],
		       n_hosts*sizeof(uint**), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(local_spike_buffer_map),
			       &d_local_spike_buffer_map, sizeof(uint***)));

#ifdef CHECKRC
  //// TEMPORARY, FOR CHECK
  std::cout << "////////////////////////////////////////\n";
  std::cout << "IN MAP CALIBRATION\n";
  
  uint tmp_n_hosts = 2;
  uint tmp_tg_host = 0;
  uint tmp_src_host = 1;
  
  uint **tmp_pt2[tmp_n_hosts];
  uint tmp_n[tmp_n_hosts];
  uint tmp_map[h_node_map_block_size];
  uint n_map;
  uint n_blocks;

  gpuErrchk(hipMemcpy(tmp_n, d_n_local_source_node_map,
		       tmp_n_hosts*sizeof(uint), hipMemcpyDeviceToHost));
  n_map = tmp_n[tmp_tg_host];
  if (n_map>0) {
    std::cout << "////////////////////////////////////////\n";
    std::cout << "Local Source Node Map\n";
    std::cout << "target host: " << tmp_tg_host << "\n";
    std::cout << "n_local_source_node_map: " << n_map << "\n";
    gpuErrchk(hipMemcpy(tmp_pt2, d_local_source_node_map,
			 tmp_n_hosts*sizeof(uint**), hipMemcpyDeviceToHost));
  
    n_blocks = (n_map - 1) / h_node_map_block_size + 1;
    std::cout << "n_blocks: " << n_blocks << "\n";
    uint *tmp_pt1[n_blocks];
    gpuErrchk(hipMemcpy(tmp_pt1, tmp_pt2[tmp_tg_host],
			 n_blocks*sizeof(uint*), hipMemcpyDeviceToHost));
    
    for (uint ib=0; ib<n_blocks; ib++) {
      std::cout << "block " << ib << "\n";
      uint n = h_node_map_block_size;
      if (ib==n_blocks-1) {
	n = (n_map - 1) % h_node_map_block_size + 1;
      }
      gpuErrchk(hipMemcpy(tmp_map, tmp_pt1[ib],
			   n*sizeof(uint), hipMemcpyDeviceToHost));
      std::cout << "local source node index\n";
      for (uint i=0; i<n; i++) {
	std::cout << tmp_map[i] << "\n";
      }
    }
  }

  //gpuErrchk(hipMemcpy(tmp_n, d_n_local_spike_buffer_map,
  gpuErrchk(hipMemcpy(tmp_n, d_n_remote_source_node_map,
		       tmp_n_hosts*sizeof(uint), hipMemcpyDeviceToHost));
  n_map = tmp_n[tmp_src_host];
  if (n_map>0) {
    std::cout << "////////////////////////////////////////\n";
    std::cout << "Local Spike Buffer Map\n";
    std::cout << "source host: " << tmp_src_host << "\n";
    std::cout << "n_local_spike_buffer_map: " << n_map << "\n";
    gpuErrchk(hipMemcpy(tmp_pt2, d_local_spike_buffer_map,
			 tmp_n_hosts*sizeof(uint**), hipMemcpyDeviceToHost));
  
    n_blocks = (n_map - 1) / h_node_map_block_size + 1;
    std::cout << "n_blocks: " << n_blocks << "\n";
    uint *tmp_pt1[n_blocks];
    gpuErrchk(hipMemcpy(tmp_pt1, tmp_pt2[tmp_src_host],
			 n_blocks*sizeof(uint*), hipMemcpyDeviceToHost));
    
    for (uint ib=0; ib<n_blocks; ib++) {
      std::cout << "block " << ib << "\n";
      uint n = h_node_map_block_size;
      if (ib==n_blocks-1) {
	n = (n_map - 1) % h_node_map_block_size + 1;
      }
      gpuErrchk(hipMemcpy(tmp_map, tmp_pt1[ib],
			   n*sizeof(uint), hipMemcpyDeviceToHost));
      std::cout << "local spike buffer index\n";
      for (uint i=0; i<n; i++) {
	std::cout << tmp_map[i] << "\n";
      }
    }
  }

  ////////////////////////////////////////
#endif

  uint n_nodes = GetNLocalNodes(); // number of nodes
  // n_target_hosts[i_node] is the number of remote target hosts
  // on which each local node
  // has outgoing connections
  // allocate d_n_target_hosts[n_nodes] and init to 0
  // std::cout << "allocate d_n_target_hosts n_nodes: " << n_nodes << "\n";
  CUDAMALLOCCTRL("&d_n_target_hosts",&d_n_target_hosts, n_nodes*sizeof(uint));
  // std::cout << "d_n_target_hosts: " << d_n_target_hosts << "\n";
  gpuErrchk(hipMemset(d_n_target_hosts, 0, n_nodes*sizeof(uint)));
  // allocate d_n_target_hosts_cumul[n_nodes+1]
  // representing the prefix scan (cumulative sum) of d_n_target_hosts
  CUDAMALLOCCTRL("&d_n_target_hosts_cumul",&d_n_target_hosts_cumul, (n_nodes+1)*sizeof(uint));

  // For each local node, count the number of remote target hosts
  // on which it has outgoing connections, i.e. n_target_hosts[i_node] 
  // Loop on target hosts
  for (int tg_host=0; tg_host<n_hosts; tg_host++) {
    if (tg_host != i_host) {
      uint **d_node_map = hd_local_source_node_map[tg_host];
      uint n_node_map = h_n_local_source_node_map[tg_host];
      // Launch kernel that searches each node in the map
      // of local source nodes having outgoing connections to target host
      // if found, increase n_target_hosts[i_node]
      searchNodeIndexInMapKernel<<<(n_nodes+1023)/1024, 1024>>>
	(d_node_map, n_node_map, d_n_target_hosts, n_nodes);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    }
  }

#ifdef CHECKRC  
  // TEMPORARY, FOR TESTING
  uint h_n_target_hosts[n_nodes];
  gpuErrchk(hipMemcpy(h_n_target_hosts, d_n_target_hosts,
  		       n_nodes*sizeof(uint), hipMemcpyDeviceToHost));
  std::cout << "////////////////////////////////////////\n";
  std::cout << "i_node, n_target_hosts\n";
  for (uint i_node=0; i_node<n_nodes; i_node++) {
    std::cout << i_node << "\t" << h_n_target_hosts[i_node] << "\n";
  }
  ////////////////////////////////////////////////
#endif
  
  //////////////////////////////////////////////////////////////////////
  // Evaluate exclusive sum of reverse connections per target node
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				d_n_target_hosts,
				d_n_target_hosts_cumul,
				n_nodes+1);
  // Allocate temporary storage
  CUDAMALLOCCTRL("&d_temp_storage",&d_temp_storage, temp_storage_bytes);
  // Run exclusive prefix sum
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
				d_n_target_hosts,
				d_n_target_hosts_cumul,
				n_nodes+1);
  CUDAFREECTRL("d_temp_storage",d_temp_storage);
  // The last element is the sum of all elements of n_target_hosts
  uint n_target_hosts_sum;
  gpuErrchk(hipMemcpy(&n_target_hosts_sum, &d_n_target_hosts_cumul[n_nodes],
		       sizeof(uint), hipMemcpyDeviceToHost));

#ifdef CHECKRC
  // TEMPORARY, FOR TESTING
  uint h_n_target_hosts_cumul[n_nodes+1];
  gpuErrchk(hipMemcpy(h_n_target_hosts_cumul, d_n_target_hosts_cumul,
  		       (n_nodes+1)*sizeof(uint), hipMemcpyDeviceToHost));
  std::cout << "////////////////////////////////////////\n";
  std::cout << "i_node, n_target_hosts_cumul\n";
  for (uint i_node=0; i_node<n_nodes+1; i_node++) {
    std::cout << i_node << "\t" << h_n_target_hosts_cumul[i_node] << "\n";
  }
  ////////////////////////////////////////////////
#endif
  
  //////////////////////////////////////////////////////////////////////
  // allocate global array with remote target hosts of all nodes
  CUDAMALLOCCTRL("&d_target_host_array",&d_target_host_array, n_target_hosts_sum*sizeof(uint));
  // allocate global array with remote target hosts map index
  CUDAMALLOCCTRL("&d_target_host_i_map",&d_target_host_i_map, n_target_hosts_sum*sizeof(uint));
  // allocate array of pointers to the starting position in target_host array
  // of the target hosts for each node
  CUDAMALLOCCTRL("&d_node_target_hosts",&d_node_target_hosts, n_nodes*sizeof(uint*));
  // allocate array of pointers to the starting position in target_host_i_map
  // of the target hosts map indexes for each node
  CUDAMALLOCCTRL("&d_node_target_host_i_map",&d_node_target_host_i_map, n_nodes*sizeof(uint*));
  // Launch kernel to evaluate the pointers d_node_target_hosts
  // and d_node_target_host_i_map from the positions in target_host_array
  // given by  n_target_hosts_cumul
  setTargetHostArrayNodePointersKernel<<<(n_nodes+1023)/1024, 1024>>>
    (d_target_host_array, d_target_host_i_map, d_n_target_hosts_cumul,
     d_node_target_hosts, d_node_target_host_i_map, n_nodes);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  // reset to 0 d_n_target_hosts[n_nodes] to reuse it in the next kernel
  gpuErrchk(hipMemset(d_n_target_hosts, 0, n_nodes*sizeof(uint)));

  // Loop on target hosts
  for (int tg_host=0; tg_host<n_hosts; tg_host++) {
    if (tg_host != i_host) {
      uint **d_node_map = hd_local_source_node_map[tg_host];
      uint n_node_map = h_n_local_source_node_map[tg_host];
      // Launch kernel to fill the arrays target_host_array
      // and target_host_i_map using the node map
      fillTargetHostArrayFromMapKernel<<<(n_nodes+1023)/1024, 1024>>>
	(d_node_map, n_node_map, d_n_target_hosts, d_node_target_hosts,
	 d_node_target_host_i_map, n_nodes, tg_host);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
    }
  }

#ifdef CHECKRC
    // TEMPORARY, FOR TESTING
  std::cout << "////////////////////////////////////////\n";
  std::cout << "Checking node_target_hosts and node_target_host_i_map\n";
  uint *hd_node_target_hosts[n_nodes];
  uint *hd_node_target_host_i_map[n_nodes];
  uint h_node_target_hosts[n_hosts];
  uint h_node_target_host_i_map[n_hosts];
  gpuErrchk(hipMemcpy(h_n_target_hosts, d_n_target_hosts,
  		       n_nodes*sizeof(uint), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(hd_node_target_hosts, d_node_target_hosts,
  		       n_nodes*sizeof(uint*), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(hd_node_target_host_i_map, d_node_target_host_i_map,
  		       n_nodes*sizeof(uint*), hipMemcpyDeviceToHost));
  for (uint i_node=0; i_node<n_nodes; i_node++) {
    std::cout << "\ni_node: " << i_node << "\n";
    uint nth = h_n_target_hosts[i_node];
    std::cout << "\tn_target_hosts: " << nth << "\n";
    
    gpuErrchk(hipMemcpy(h_node_target_hosts, hd_node_target_hosts[i_node],
			 nth*sizeof(uint), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_node_target_host_i_map,
			 hd_node_target_host_i_map[i_node],
			 nth*sizeof(uint), hipMemcpyDeviceToHost));

    std::cout << "node_target_hosts\tnode_target_host_i_map\n";
    for (int ith=0; ith<nth; ith++) {
      std::cout << h_node_target_hosts[ith] << "\t"
		<< h_node_target_host_i_map[ith] << "\n";
    }
  }
#endif

  
  return 0;
}

      

// kernel that counts source nodes actually used in new connections
__global__ void countUsedSourceNodeKernel(uint n_source,
					  uint *n_used_source_nodes,
					  uint *source_node_flag)
{
  uint i_source = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_source>=n_source) return;
  // Count how many source_node_flag are true using atomic increase
  // on n_used_source_nodes
  if (source_node_flag[i_source] != 0) {
    atomicAdd(n_used_source_nodes, 1);
  }
}


// device function that checks if an int value is in a sorted 2d-array 
// assuming that the entries in the 2d-array are sorted.
// The 2d-array is divided in noncontiguous blocks of size block_size
__device__ bool checkIfValueIsIn2DArr(uint value, uint **arr, uint n_elem,
				      uint block_size, uint *i_block,
				      uint *i_in_block)
{
  // If the array is empty surely the value is not contained in it
  if (n_elem<=0) {
    return false;
  }
  // determine number of blocks in array
  uint n_blocks = (n_elem - 1) / block_size + 1;
  // determine number of elements in last block
  uint n_last = (n_elem - 1) % block_size + 1;
  // check if value is between the minimum and the maximum in the map
  if (value<arr[0][0] ||
      value>arr[n_blocks-1][n_last-1]) {
    return false;
  }
  for (uint ib=0; ib<n_blocks; ib++) {
    if (arr[ib][0] > value) { // the array is sorted, so in this case
      return false;           // value cannot be in the following elements
    }
    uint n = block_size;
    if (ib==n_blocks-1) { // the last block can be not completely full
      n = n_last;
    }
    // search value in the block
    int pos = locate<uint, int>(value, arr[ib], (int)n);
    // if value is in the block return true
    if (pos>=0 && pos<n && arr[ib][pos]==value) {
      *i_block = ib;
      *i_in_block = pos;
      return true;
    }
  }
  return false; // value not found
}  


// kernel that searches node indexes in map
// increase counter of mapped nodes
__global__ void searchNodeIndexInMapKernel
(
 uint **node_map,
 uint n_node_map,
 uint *count_mapped, // i.e. *n_target_hosts for our application
 uint n_node)
{
  uint i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node>=n_node) return;
  uint i_block;
  uint i_in_block;
  // check if node index is in map
  bool mapped = checkIfValueIsIn2DArr(i_node, node_map,
				      n_node_map, node_map_block_size,
				      &i_block, &i_in_block);
  // If it is mapped
  if (mapped) {
    // i_node_map = i_block*node_map_block_size + i_in_block;
    count_mapped[i_node]++;
  }
}

// kernel that searches node indexes not in map
// flags nodes not yet mapped and counts them
__global__ void searchNodeIndexNotInMapKernel
(
 uint **node_map,
 uint n_node_map,
 uint *sorted_node_index,
 bool *node_to_map,
 uint *n_node_to_map,
 uint n_node)
{
  uint i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node>=n_node) return;
  // Check for sorted_node_index unique values:
  // - either if it is the first of the array (i_node = 0)
  // - or it is different from previous
  uint node_index = sorted_node_index[i_node];
  if (i_node==0 || node_index!=sorted_node_index[i_node-1]) {
    uint i_block;
    uint i_in_block;
    bool mapped = checkIfValueIsIn2DArr(node_index, node_map,
					n_node_map, node_map_block_size,
					&i_block, &i_in_block);
    // If it is not in the map then flag it to be mapped
    // and atomic increase n_new_source_node_map
    if (!mapped) {
      node_to_map[i_node] = true;
      atomicAdd(n_node_to_map, 1);
    }
  }
}


// kernel that checks if nodes are already in map
// if not insert them in the map
// In the target host unmapped remote source nodes must be mapped
// to local nodes from n_nodes to n_nodes + n_node_to_map
__global__ void insertNodesInMapKernel
(
 uint **node_map,
 uint **spike_buffer_map,
 uint spike_buffer_map_i0,
 uint old_n_node_map,
 uint *sorted_node_index,
 bool *node_to_map,
 uint *i_node_to_map,
 uint n_node)
{
  uint i_node = threadIdx.x + blockIdx.x * blockDim.x;
  // if thread is out of range or node is already mapped, return
  if (i_node>=n_node || !node_to_map[i_node]) return;
  // node has to be inserted in the map
  // get and atomically increase index of node to be mapped
  uint pos = atomicAdd(i_node_to_map, 1);
  uint i_node_map = old_n_node_map + pos;
  uint i_block = i_node_map / node_map_block_size;
  uint i = i_node_map % node_map_block_size;
  node_map[i_block][i] = sorted_node_index[i_node];
  if (spike_buffer_map != NULL) {
    spike_buffer_map[i_block][i] = spike_buffer_map_i0 + pos;
  }
}



__global__ void MapIndexToSpikeBufferKernel(uint n_hosts, uint *host_offset,
					    uint *node_index)
{
  const uint i_host = blockIdx.x;
  if (i_host < n_hosts) {    
    const uint pos = host_offset[i_host];
    const uint num = host_offset[i_host+1] - pos;
    for (uint i_elem = threadIdx.x; i_elem < num; i_elem += blockDim.x) {
      const uint i_node_map = node_index[pos + i_elem];
      const uint i_block = i_node_map / node_map_block_size;
      const uint i = i_node_map % node_map_block_size;
      const uint i_spike_buffer = local_spike_buffer_map[i_host][i_block][i];
      node_index[pos + i_elem] = i_spike_buffer; 
    }
  }
}


__global__ void addOffsetToSpikeBufferMapKernel(uint i_host, uint n_node_map,
						uint i_image_node_0)
{
  uint i_node_map = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node_map>=n_node_map) return;
  
  const uint i_block = i_node_map / node_map_block_size;
  const uint i = i_node_map % node_map_block_size;
  local_spike_buffer_map[i_host][i_block][i] += i_image_node_0;
}

int NESTGPU::addOffsetToSpikeBufferMap()
{
  uint i_image_node_0 = GetNLocalNodes();

  for (int i_host=0; i_host<n_hosts_; i_host++) {
    if (i_host != this_host_) {
      uint n_node_map = h_n_remote_source_node_map[i_host];
      if (n_node_map > 0) {
	addOffsetToSpikeBufferMapKernel<<<(n_node_map+1023)/1024, 1024>>>
	  (i_host, n_node_map, i_image_node_0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
      }
    }
  }
    
  return 0;
}
