#include "hip/hip_runtime.h"
/*
Copyright (C) 2016 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include "send_spike.h"
#include "spike_buffer.h"
#include "rk5.h"

extern __device__ int Aeif_i_node_0; 

__device__ double *GetSpikeArray;

double *d_GetSpikeArray;

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ int locate(uint val, uint *data, int n)
{
// remember to add a check that data[0]<=val<=data[n-1]
// what to do if not?
   int i_left = 0;
   int i_right = n-1;
   int i = (i_left+i_right)/2;
   while(i_right-i_left>1) {
      if (data[i] > val) i_right = i;
      else if (data[i]<val) i_left = i;
      else break;
      i=(i_left+i_right)/2;
   }

   return i;
}

__global__ void CollectSpikes(uint n_get_spikes, int nvar, int nparams)
{
  uint blockId   = blockIdx.y * gridDim.x + blockIdx.x;				
  uint array_idx = blockId * blockDim.x + threadIdx.x; 
  if (array_idx<n_get_spikes) {

    int i_spike = locate(array_idx, SpikeTargetNumSum, (*SpikeNum) + 1);
    int i_syn = array_idx - SpikeTargetNumSum[i_spike];
    int i_source = SpikeSourceIdx[i_spike];
    int i_conn = SpikeConnIdx[i_spike];
    float height = SpikeHeight[i_spike];
    int i_target = ConnectionGroupTargetNeuron[i_conn*NSpikeBuffer+i_source]
      [i_syn] - Aeif_i_node_0;
    unsigned char i_port = ConnectionGroupTargetPort[i_conn*NSpikeBuffer
						     +i_source][i_syn];
    float weight = ConnectionGroupTargetWeight[i_conn*NSpikeBuffer+i_source]
      [i_syn];
    
    // printf("handles spike %d src %d conn %d syn %d target %d"
    // " port %d weight %f\n",
    // i_spike, i_source, i_conn, i_syn, i_target,
    // i_port, weight);

    // IMPROVE THIS PART
    /////////////////////////////////////////////////////////////////
    int i = i_port*ARRAY_SIZE + i_target;
    int j = (N0_PARAMS + 3 + 4*i_port)*ARRAY_SIZE + i_target; // g0(i)
    double d_val = (double)(height*weight*ParamsArr[j]);
    atomicAddDouble(&GetSpikeArray[i], d_val); 
    ////////////////////////////////////////////////////////////////
  }
}

// improve using a grid
__global__ void GetSpikes(int n_ports)
{
  int i_array = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_array < ARRAY_SIZE*n_ports) {
     int i_target = i_array % ARRAY_SIZE;
     int i_port = i_array / ARRAY_SIZE;
     int i = (N0_VAR + 1 + 2*i_port)*ARRAY_SIZE + i_target; // g1(i)
     double d_val = GetSpikeArray[i_array] + (double)YArr[i];  
     YArr[i] = (float)d_val;
  }
}
    // REMOVE THIS PART
    /////////////////////////////////////////////////////////////////
 //   int i = (N0_VAR + 1 + 2*i_port)*ARRAY_SIZE + i_target; // g1(i)
 //   int j = (N0_PARAMS + 3 + 4*i_port)*ARRAY_SIZE + i_target; // g0(i)
 //   atomicAdd(&YArr[i], height*weight*ParamsArr[j]); 
    ////////////////////////////////////////////////////////////////

__global__
void DeviceInitGetSpikeArray(double *get_spike_array)
{
  GetSpikeArray = get_spike_array;
}

int InitGetSpikeArray(int n_neurons, int n_ports)
{
  gpuErrchk(hipMalloc(&d_GetSpikeArray, n_neurons*n_ports*sizeof(double)));
  DeviceInitGetSpikeArray<<<1, 1>>>(d_GetSpikeArray);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  return 0;
}

int ClearGetSpikeArray(int n_neurons, int n_ports)
{
  gpuErrchk(hipMemset(d_GetSpikeArray, 0, n_neurons*n_ports*sizeof(double)));

  return 0;
}

int FreeGetSpikeArray()
{
  gpuErrchk(hipFree(d_GetSpikeArray));

  return 0;
}

