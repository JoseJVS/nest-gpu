#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>
#include <stdio.h>
#include <iostream>
#include "ngpu_exception.h"
#include "cuda_error.h"
#include "stdp.h"

using namespace stdp_ns;

__device__ void STDPUpdate(float *weight_pt, float Dt, float *param)
{
  float tau_plus = param[i_tau_plus];
  float tau_minus = param[i_tau_minus];
  float Wplus = param[i_Wplus];
  float alpha = param[i_alpha];
  float mu_plus = param[i_mu_plus];
  float mu_minus = param[i_mu_minus];
  float Wmax = param[i_Wmax];

  float w = *weight_pt;
  if (Dt>=0) {
    float fact = Wplus*exp(-Dt/tau_plus);
    float w1 = w + fact*pow(1.0 - w/Wmax, mu_plus);
    *weight_pt = w1 < Wmax ? w1 : Wmax;
  }
  else {
    float fact = -alpha*Wplus*exp(Dt/tau_minus);
    float w1 = w + fact*pow(w/Wmax, mu_minus);
    *weight_pt =     *weight_pt = w1 >0.0 ? w1 : 0.0;
  }
}

int STDP::Init()
{
  type_ = i_stdp_model;
  n_param_ = N_PARAM;
  param_name_ = stdp_param_name;
  gpuErrchk(hipMalloc(&d_param_arr_, n_param_*sizeof(float)));
  SetParam("tau_plus", 20.0);
  SetParam("tau_minus", 20.0);
  SetParam("Wplus", 0.01);
  SetParam("alpha", 1.0);
  SetParam("mu_plus", 1.0);
  SetParam("mu_minus", 1.0);
  SetParam("Wmax", 100.0);

  return 0;
}
