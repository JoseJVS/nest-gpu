#include "hip/hip_runtime.h"
/*
Copyright (C) 2016 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "poisson.h"
#include "spike_buffer.h"
#include "cuda_error.h"

__device__ unsigned int *PoissonData;

__global__ void PoissonUpdate(unsigned int *poisson_data)
{
  PoissonData = poisson_data;
}

__global__
void PoissonSendSpikes(int i_node_0, int n_nodes)
{
  int i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node < n_nodes) {
    int i_node_abs = i_node_0 + i_node;
    unsigned int height = PoissonData[i_node];
    if (height>0) {
      PushSpike(i_node_abs, (float)height);
    }
  }
}

__global__
void FixPoissonGenerator(unsigned int *poisson_data, int n, float mean)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    unsigned int val = poisson_data[i];
    if (val>mean*5) {
      poisson_data[i] =0;
    }
  }
}

int PoissonGenerator::Init(hiprandGenerator_t *random_generator, unsigned int n)
{
  poisson_data_size_ = n;
  // Allocate n integers on device
  CUDA_CALL(hipMalloc((void **)&dev_poisson_data_, n * sizeof(unsigned int)));
  random_generator_ = random_generator;

  return 0;
}

int PoissonGenerator::Generate()
{
  return Generate(n_steps_);
}

int PoissonGenerator::Generate(int max_n_steps)
{
  if (max_n_steps <= 0) {
    more_steps_ = n_steps_;
  }
  else {
    more_steps_ = min(n_steps_, max_n_steps);
  }
  // Generate N floats on device
  CURAND_CALL(hiprandGeneratePoisson(*random_generator_, dev_poisson_data_,
				    n_nodes_*more_steps_, lambda_));
  hipDeviceSynchronize();
  FixPoissonGenerator<<<(n_nodes_+1023)/1024, 1024>>>
    (dev_poisson_data_,n_nodes_*more_steps_, lambda_);
  hipDeviceSynchronize();

  return 0;
}

int PoissonGenerator::Free()
{
  CUDA_CALL(hipFree(dev_poisson_data_));

  return 0;
}

PoissonGenerator::~PoissonGenerator()
{
  //Free();
}

PoissonGenerator::PoissonGenerator()
{
  buffer_size_ = 100000;
  n_nodes_ = 0;
}

int PoissonGenerator::Create(hiprandGenerator_t *random_generator,
			     int i_node_0, int n_nodes, float lambda)
{
  i_node_0_ = i_node_0;
  n_nodes_ = n_nodes;
  lambda_ = lambda;
  
  n_steps_ = (buffer_size_ - 1)/n_nodes + 1;
  // with the above formula:
  // buffer_size <= n_nodes*n_steps <= buffer_size + n_nodes - 1
  Init(random_generator, n_nodes_*n_steps_);
  i_step_ = 0;
       
  return 0;
}

int PoissonGenerator::Update(int max_n_steps)
{
  if (i_step_ == 0) {
    Generate(max_n_steps);
  }

  if (i_step_ == more_steps_) {
    fprintf(stderr, "Step index larger than maximum number of steps in poisson"
	    " generator\n");
    exit(0);
  }
  
  PoissonUpdate<<<1, 1>>>(&dev_poisson_data_[i_step_*n_nodes_]);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  PoissonSendSpikes<<<(n_nodes_+1023)/1024, 1024>>>(i_node_0_, n_nodes_);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  i_step_++;
  if (i_step_ == n_steps_) i_step_ = 0;

  return 0;
}

