#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>
#include <stdio.h>
#include "send_spike.h"
#include "cuda_error.h"

int *d_SpikeNum;
int *d_SpikeSourceIdx;
int *d_SpikeConnIdx;
float *d_SpikeHeight;
int *d_SpikeTargetNum;

__device__ int MaxSpikeNum;
__device__ int *SpikeNum;
__device__ int *SpikeSourceIdx;
__device__ int *SpikeConnIdx;
__device__ float *SpikeHeight;
__device__ int *SpikeTargetNum;

__device__ void SendSpike(int i_source, int i_conn, float height,
			  int target_num)
{
  int pos = atomicAdd(SpikeNum, 1);
  SpikeSourceIdx[pos] = i_source;
  SpikeConnIdx[pos] = i_conn;
  SpikeHeight[pos] = height;
  SpikeTargetNum[pos] = target_num;
}

__global__ void DeviceSpikeInit(int *spike_num, int *spike_source_idx,
				int *spike_conn_idx, float *spike_height,
				int *spike_target_num,
				int max_spike_num)
{
  SpikeNum = spike_num;
  SpikeSourceIdx = spike_source_idx;
  SpikeConnIdx = spike_conn_idx;
  SpikeHeight = spike_height;
  SpikeTargetNum = spike_target_num;
  MaxSpikeNum = max_spike_num;
  *SpikeNum = 0;
}


void SpikeInit(int max_spike_num)
{
  //h_SpikeTargetNum = new int[PrefixScan::AllocSize];

  gpuErrchk(hipMalloc(&d_SpikeNum, sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeSourceIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeConnIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeHeight, max_spike_num*sizeof(float)));
  gpuErrchk(hipMalloc(&d_SpikeTargetNum, max_spike_num*sizeof(int)));
  //printf("here: SpikeTargetNum size: %d", max_spike_num);
  DeviceSpikeInit<<<1,1>>>(d_SpikeNum, d_SpikeSourceIdx, d_SpikeConnIdx,
			   d_SpikeHeight, d_SpikeTargetNum, max_spike_num);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

__global__ void SpikeReset()
{
  *SpikeNum = 0;
}
