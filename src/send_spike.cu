#include "hip/hip_runtime.h"
/*
Copyright (C) 2016 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include "send_spike.h"
#include "cuda_error.h"

int *d_SpikeNum;
int *d_SpikeSourceIdx;
int *d_SpikeConnIdx;
float *d_SpikeHeight;
int *d_SpikeTargetNum;

//int *h_SpikeSourceIdx;
//int *h_SpikeConnIdx;
//float *h_SpikeHeight;
//int *h_SpikeTargetNum;

__device__ int MaxSpikeNum;
__device__ int *SpikeNum;
__device__ int *SpikeSourceIdx;
__device__ int *SpikeConnIdx;
__device__ float *SpikeHeight;
__device__ int *SpikeTargetNum;

__device__ void SendSpike(int i_source, int i_conn, float height,
			  int target_num)
{
  int pos = atomicAdd(SpikeNum, 1);
  SpikeSourceIdx[pos] = i_source;
  SpikeConnIdx[pos] = i_conn;
  SpikeHeight[pos] = height;
  SpikeTargetNum[pos] = target_num;
  //printf("ss: %d\t%d\t%d\n", i_source, i_conn, target_num);
  //printf("sn: %d\n", *SpikeNum);
  //printf("Pos: %d\t ssi: %d\n", pos, SpikeSourceIdx[pos]);
    
}

__global__ void DeviceSpikeInit(int *spike_num, int *spike_source_idx,
				int *spike_conn_idx, float *spike_height,
				int *spike_target_num,
				int max_spike_num)
{
  SpikeNum = spike_num;
  SpikeSourceIdx = spike_source_idx;
  SpikeConnIdx = spike_conn_idx;
  SpikeHeight = spike_height;
  SpikeTargetNum = spike_target_num;
  MaxSpikeNum = max_spike_num;
  *SpikeNum = 0;
}


void SpikeInit(int max_spike_num)
{
  //h_SpikeSourceIdx = new int[max_spike_num];
  //h_SpikeConnIdx = new int[max_spike_num];
  //h_SpikeHeight = new float[max_spike_num];
  //h_SpikeTargetNum = new int[PrefixScan::AllocSize];

  gpuErrchk(hipMalloc(&d_SpikeNum, sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeSourceIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeConnIdx, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_SpikeHeight, max_spike_num*sizeof(float)));
  gpuErrchk(hipMalloc(&d_SpikeTargetNum, max_spike_num*sizeof(int)));

  DeviceSpikeInit<<<1,1>>>(d_SpikeNum, d_SpikeSourceIdx, d_SpikeConnIdx,
			   d_SpikeHeight, d_SpikeTargetNum, max_spike_num);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
}

__global__ void SpikeReset()
{
  *SpikeNum = 0;
}
