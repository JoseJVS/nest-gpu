#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include "ngpu_exception.h"
#include "cuda_error.h"
#include "base_neuron.h"
#include "spike_buffer.h"
__global__ void BaseNeuronSetFloatArray(float *arr, int n_elems, int step,
					float val)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<n_elems) {
    arr[array_idx*step] = val;
  }
}

__global__ void BaseNeuronSetFloatPtArray(float *arr, int *pos, int n_elems,
					  int step, float val)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<n_elems) {
    arr[pos[array_idx]*step] = val;
  }
}

int BaseNeuron::Init(int i_node_0, int n_nodes, int n_ports,
		     int i_group, unsigned long long *seed)
{
  node_type_= 0; // NULL MODEL
  i_node_0_ = i_node_0;
  n_nodes_ = n_nodes;
  n_ports_ = n_ports;
  i_group_ = i_group;
  seed_ = seed;
  
  n_scal_var_ = 0;
  n_vect_var_ = 0;
  n_scal_params_ = 0;
  n_vect_params_ = 0;
  n_var_ = 0;
  n_params_ = 0;

  get_spike_array_ = NULL;
  port_weight_arr_ = NULL;
  port_weight_arr_step_ = 0;
  port_weight_port_step_ = 0;
  port_input_arr_ = NULL;
  port_input_arr_step_ = 0;
  port_input_port_step_ = 0;
  var_arr_ = NULL;
  params_arr_ = NULL;
  scal_var_name_ = NULL;
  vect_var_name_= NULL;
  scal_param_name_ = NULL;
  vect_param_name_ = NULL;
  d_dir_conn_array_ = NULL;
  n_dir_conn_ = 0;
  has_dir_conn_ = false;
 
  return 0;
}			    

int BaseNeuron::SetScalParam(int i_neuron, int n_neurons,
			     std::string param_name, float val)
{
  if (!IsScalParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized scalar parameter ")
			 + param_name);
  }
  CheckNeuronIdx(i_neuron);
  CheckNeuronIdx(i_neuron + n_neurons - 1);
  float *param_pt = GetParamPt(i_neuron, param_name);
  BaseNeuronSetFloatArray<<<(n_neurons+1023)/1024, 1024>>>
    (param_pt, n_neurons, n_params_, val);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  
  return 0;
}

int BaseNeuron::SetScalParam( int *i_neuron, int n_neurons,
			      std::string param_name, float val)
{
  if (!IsScalParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized scalar parameter ")
				     + param_name);
  }
  int *d_i_neuron;
  gpuErrchk(hipMalloc(&d_i_neuron, n_neurons*sizeof(int)));
  gpuErrchk(hipMemcpy(d_i_neuron, i_neuron, n_neurons*sizeof(int),
		       hipMemcpyHostToDevice));
  float *param_pt = GetParamPt(0, param_name);
  BaseNeuronSetFloatPtArray<<<(n_neurons+1023)/1024, 1024>>>
    (param_pt, d_i_neuron, n_neurons, n_params_, val);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  gpuErrchk(hipFree(d_i_neuron));
  
  return 0;
}

int BaseNeuron::SetVectParam( int i_neuron, int n_neurons,
			      std::string param_name, float *params,
			      int vect_size)
{
  if (!IsVectParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  CheckNeuronIdx(i_neuron);
  CheckNeuronIdx(i_neuron + n_neurons - 1);
  if (vect_size != n_ports_) {
    throw ngpu_exception("Parameter vector size must be equal "
			 "to the number of ports.");
  }
  float *param_pt;
    
  for (int i_vect=0; i_vect<vect_size; i_vect++) {
    param_pt = GetParamPt(i_neuron, param_name, i_vect);
    BaseNeuronSetFloatArray<<<(n_neurons+1023)/1024, 1024>>>
      (param_pt, n_neurons, n_params_, params[i_vect]);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  return 0;
}

int BaseNeuron::SetVectParam( int *i_neuron, int n_neurons,
			      std::string param_name, float *params,
			      int vect_size)
{
  if (!IsVectParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  if (vect_size != n_ports_) {
    throw ngpu_exception("Parameter vector size must be equal "
			 "to the number of ports.");
  }
  int *d_i_neuron;
  gpuErrchk(hipMalloc(&d_i_neuron, n_neurons*sizeof(int)));
  gpuErrchk(hipMemcpy(d_i_neuron, i_neuron, n_neurons*sizeof(int),
		       hipMemcpyHostToDevice));
  for (int i_vect=0; i_vect<vect_size; i_vect++) {
    float *param_pt = GetParamPt(0, param_name, i_vect);
    BaseNeuronSetFloatPtArray<<<(n_neurons+1023)/1024, 1024>>>
      (param_pt, d_i_neuron, n_neurons, n_params_, params[i_vect]);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  gpuErrchk(hipFree(d_i_neuron));

  return 0;
}

int BaseNeuron::GetScalVarIdx(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_scal_var_; i_var++) {
    if (var_name == scal_var_name_[i_var]) break;
  }
  if (i_var == n_scal_var_) {
    throw ngpu_exception(std::string("Unrecognized scalar variable ")
			 + var_name);
  }
  
  return i_var;
}

int BaseNeuron::GetVectVarIdx(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_vect_var_; i_var++) {
    if (var_name == vect_var_name_[i_var]) break;
  }
  if (i_var == n_vect_var_) {
    throw ngpu_exception(std::string("Unrecognized vector variable ")
				     + var_name);
  }
  
  return i_var;
}

int BaseNeuron::GetScalParamIdx(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_scal_params_; i_param++) {
    if (param_name == scal_param_name_[i_param]) break;
  }
  if (i_param == n_scal_params_) {
    throw ngpu_exception(std::string("Unrecognized parameter ")
			 + param_name);
  }
  
  return i_param;
}

int BaseNeuron::GetVectParamIdx(std::string param_name)
{  
  int i_param;
  for (i_param=0; i_param<n_vect_params_; i_param++) {
    if (param_name == vect_param_name_[i_param]) break;
  }
  if (i_param == n_vect_params_) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  
  return i_param;
}

float *BaseNeuron::GetVarArr()
{
  return var_arr_;
}

float *BaseNeuron::GetParamArr()
{
  return params_arr_;
}

bool BaseNeuron::IsScalVar(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_scal_var_; i_var++) {
    if (var_name == scal_var_name_[i_var]) return true;
  }
  return false;
}

bool BaseNeuron::IsVectVar(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_vect_var_; i_var++) {
    if (var_name == vect_var_name_[i_var]) return true;
  }
  return false;
}

bool BaseNeuron::IsScalParam(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_scal_params_; i_param++) {
    if (param_name == scal_param_name_[i_param]) return true;
  }
  return false;
}

bool BaseNeuron::IsVectParam(std::string param_name)
{  
  int i_param;
  for (i_param=0; i_param<n_vect_params_; i_param++) {
    if (param_name == vect_param_name_[i_param]) return true;
  }
  return false;
}

int BaseNeuron::CheckNeuronIdx(int i_neuron)
{
  if (i_neuron>=n_nodes_) {
    throw ngpu_exception("Neuron index must be lower then n. of neurons");
  }
  else if (i_neuron<0) {
    throw ngpu_exception("Neuron index must be >= 0");
  }
  return 0;
}

int BaseNeuron::CheckPortIdx(int i_port)
{
  if (i_port>=n_ports_) {
    throw ngpu_exception("Port index must be lower then n. of ports");
  }
  else if (i_port<0) {
    throw ngpu_exception("Port index must be >= 0");
  }
  return 0;
}

float *BaseNeuron::GetVarPt(int i_neuron, std::string var_name,
			    int i_port /*=0*/)
{
  CheckNeuronIdx(i_neuron);
  if (i_port!=0) {
    CheckPortIdx(i_port);
  }
    
  if (IsScalVar(var_name)) {
    int i_var =  GetScalVarIdx(var_name);
    return GetVarArr() + i_neuron*n_var_ + i_var;
  }
  else if (IsVectVar(var_name)) {
    int i_vvar =  GetVectVarIdx(var_name);
    return GetVarArr() + i_neuron*n_var_ + n_scal_var_
      + i_port*n_vect_var_ + i_vvar;
  }
  else {
    throw ngpu_exception(std::string("Unrecognized variable ")
			 + var_name);
  }
}

float *BaseNeuron::GetParamPt(int i_neuron, std::string param_name,
			      int i_port /*=0*/)
{
  CheckNeuronIdx(i_neuron);
  if (i_port!=0) {
    CheckPortIdx(i_port);
  }
  if (IsScalParam(param_name)) {
    int i_param =  GetScalParamIdx(param_name);
    return GetParamArr() + i_neuron*n_params_ + i_param;
  }
  else if (IsVectParam(param_name)) {
    int i_vparam =  GetVectParamIdx(param_name);
    return GetParamArr() + i_neuron*n_params_ + n_scal_params_
      + i_port*n_vect_params_ + i_vparam;
  }
  else {
    throw ngpu_exception(std::string("Unrecognized parameter ")
			 + param_name);
  }
}

float BaseNeuron::GetSpikeActivity(int i_neuron)
{
  int i_spike_buffer = i_neuron + i_node_0_;
  int Ns;
  gpuErrchk(hipMemcpy(&Ns, d_SpikeBufferSize + i_spike_buffer,
		       sizeof(int), hipMemcpyDeviceToHost));
  if (Ns==0) {
    return 0.0;
  }
  int time_idx;
  // get first (most recent) spike from buffer
  gpuErrchk(hipMemcpy(&time_idx, d_SpikeBufferTimeIdx + i_spike_buffer,
		       sizeof(int), hipMemcpyDeviceToHost));
  if (time_idx!=0) { // neuron is not spiking now
    return 0.0;
  }
  float spike_height;
  gpuErrchk(hipMemcpy(&spike_height, d_SpikeBufferHeight + i_spike_buffer,
		       sizeof(float), hipMemcpyDeviceToHost));

  return spike_height;
}

