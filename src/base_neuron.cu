#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include "ngpu_exception.h"
#include "cuda_error.h"
#include "base_neuron.h"
#include "spike_buffer.h"
__global__ void BaseNeuronSetFloatArray(float *arr, int n_elems, int step,
					float val)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<n_elems) {
    arr[array_idx*step] = val;
  }
}

__global__ void BaseNeuronSetFloatPtArray(float *arr, int *pos, int n_elems,
					  int step, float val)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<n_elems) {
    arr[pos[array_idx]*step] = val;
  }
}

int BaseNeuron::Init(int i_node_0, int n_nodes, int n_ports,
		   int i_group)
{
  node_type_= 0; // NULL MODEL
  i_node_0_ = i_node_0;
  n_nodes_ = n_nodes;
  n_ports_ = n_ports;
  i_group_ = i_group;
  get_spike_array_ = NULL;

  return 0;
}			    

int BaseNeuron::SetScalParam(std::string param_name, int i_neuron,
		    int n_neurons, float val) {
  if (!IsScalParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized scalar parameter ")
			 + param_name);
  }
  CheckNeuronIdx(i_neuron);
  CheckNeuronIdx(i_neuron + n_neurons - 1);
  float *param_pt = GetParamPt(param_name, i_neuron);
  BaseNeuronSetFloatArray<<<(n_neurons+1023)/1024, 1024>>>
    (param_pt, n_neurons, n_params_, val);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  
  return 0;
}

int BaseNeuron::SetScalParam(std::string param_name, int *i_neuron,
		    int n_neurons, float val) {
  if (!IsScalParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized scalar parameter ")
				     + param_name);
  }
  int *d_i_neuron;
  gpuErrchk(hipMalloc(&d_i_neuron, n_neurons*sizeof(int)));
  gpuErrchk(hipMemcpy(d_i_neuron, i_neuron, n_neurons*sizeof(int),
		       hipMemcpyHostToDevice));
  float *param_pt = GetParamPt(param_name, 0);
  BaseNeuronSetFloatPtArray<<<(n_neurons+1023)/1024, 1024>>>
    (param_pt, d_i_neuron, n_neurons, n_params_, val);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  gpuErrchk(hipFree(d_i_neuron));
  
  return 0;
}

int BaseNeuron::SetVectParam(std::string param_name, int i_neuron,
			      int n_neurons, float *params, int vect_size) {
  if (!IsVectParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  CheckNeuronIdx(i_neuron);
  CheckNeuronIdx(i_neuron + n_neurons - 1);
  if (vect_size != n_ports_) {
    throw ngpu_exception("Parameter vector size must be equal "
			 "to the number of ports.");
  }
  float *param_pt;
    
  for (int i_vect=0; i_vect<vect_size; i_vect++) {
    param_pt = GetParamPt(param_name, i_neuron, i_vect);
    BaseNeuronSetFloatArray<<<(n_neurons+1023)/1024, 1024>>>
      (param_pt, n_neurons, n_params_, params[i_vect]);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  return 0;
}

int BaseNeuron::SetVectParam(std::string param_name, int *i_neuron,
			      int n_neurons, float *params, int vect_size) {
  if (!IsVectParam(param_name)) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  if (vect_size != n_ports_) {
    throw ngpu_exception("Parameter vector size must be equal "
			 "to the number of ports.");
  }
  int *d_i_neuron;
  gpuErrchk(hipMalloc(&d_i_neuron, n_neurons*sizeof(int)));
  gpuErrchk(hipMemcpy(d_i_neuron, i_neuron, n_neurons*sizeof(int),
		       hipMemcpyHostToDevice));
  for (int i_vect=0; i_vect<vect_size; i_vect++) {
    float *param_pt = GetParamPt(param_name, 0, i_vect);
    BaseNeuronSetFloatPtArray<<<(n_neurons+1023)/1024, 1024>>>
      (param_pt, d_i_neuron, n_neurons, n_params_, params[i_vect]);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  gpuErrchk(hipFree(d_i_neuron));

  return 0;
}

int BaseNeuron::GetScalVarIdx(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_scal_var_; i_var++) {
    if (var_name == scal_var_name_[i_var]) break;
  }
  if (i_var == n_scal_var_) {
    throw ngpu_exception(std::string("Unrecognized scalar variable ")
			 + var_name);
  }
  
  return i_var;
}

int BaseNeuron::GetVectVarIdx(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_vect_var_; i_var++) {
    if (var_name == vect_var_name_[i_var]) break;
  }
  if (i_var == n_vect_var_) {
    throw ngpu_exception(std::string("Unrecognized vector variable ")
				     + var_name);
  }
  
  return i_var;
}

int BaseNeuron::GetScalParamIdx(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_scal_params_; i_param++) {
    if (param_name == scal_param_name_[i_param]) break;
  }
  if (i_param == n_scal_params_) {
    throw ngpu_exception(std::string("Unrecognized parameter ")
			 + param_name);
  }
  
  return i_param;
}

int BaseNeuron::GetVectParamIdx(std::string param_name)
{  
  int i_param;
  for (i_param=0; i_param<n_vect_params_; i_param++) {
    if (param_name == vect_param_name_[i_param]) break;
  }
  if (i_param == n_vect_params_) {
    throw ngpu_exception(std::string("Unrecognized vector parameter ")
			 + param_name);
  }
  
  return i_param;
}

float *BaseNeuron::GetVarArr()
{
  return var_arr_;
}

float *BaseNeuron::GetParamArr()
{
  return params_arr_;
}

bool BaseNeuron::IsScalVar(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_scal_var_; i_var++) {
    if (var_name == scal_var_name_[i_var]) return true;
  }
  return false;
}

bool BaseNeuron::IsVectVar(std::string var_name)
{
  int i_var;
  for (i_var=0; i_var<n_vect_var_; i_var++) {
    if (var_name == vect_var_name_[i_var]) return true;
  }
  return false;
}

bool BaseNeuron::IsScalParam(std::string param_name)
{
  int i_param;
  for (i_param=0; i_param<n_scal_params_; i_param++) {
    if (param_name == scal_param_name_[i_param]) return true;
  }
  return false;
}

bool BaseNeuron::IsVectParam(std::string param_name)
{  
  int i_param;
  for (i_param=0; i_param<n_vect_params_; i_param++) {
    if (param_name == vect_param_name_[i_param]) return true;
  }
  return false;
}

int BaseNeuron::CheckNeuronIdx(int i_neuron)
{
  if (i_neuron>=n_nodes_) {
    throw ngpu_exception("Neuron index must be lower then n. of neurons");
  }
  else if (i_neuron<0) {
    throw ngpu_exception("Neuron index must be >= 0");
  }
  return 0;
}

int BaseNeuron::CheckPortIdx(int i_port)
{
  if (i_port>=n_ports_) {
    throw ngpu_exception("Port index must be lower then n. of ports");
  }
  else if (i_port<0) {
    throw ngpu_exception("Port index must be >= 0");
  }
  return 0;
}

float *BaseNeuron::GetVarPt(std::string var_name, int i_neuron,
			    int i_port /*=0*/)
{
  CheckNeuronIdx(i_neuron);
  CheckPortIdx(i_port);
    
  if (IsScalVar(var_name)) {
    int i_var =  GetScalVarIdx(var_name);
    return GetVarArr() + i_neuron*n_var_ + i_var;
  }
  else if (IsVectVar(var_name)) {
    int i_vvar =  GetVectVarIdx(var_name);
    return GetVarArr() + i_neuron*n_var_ + n_scal_var_
      + i_port*n_vect_var_ + i_vvar;
  }
  else {
    throw ngpu_exception(std::string("Unrecognized variable ")
			 + var_name);
  }
}

float *BaseNeuron::GetParamPt(std::string param_name, int i_neuron,
			      int i_port /*=0*/)
{
  CheckNeuronIdx(i_neuron);
  CheckPortIdx(i_port);
    
  if (IsScalParam(param_name)) {
    int i_param =  GetScalParamIdx(param_name);
    return GetParamArr() + i_neuron*n_params_ + i_param;
  }
  else if (IsVectParam(param_name)) {
    int i_vparam =  GetVectParamIdx(param_name);
    return GetParamArr() + i_neuron*n_params_ + n_scal_params_
      + i_port*n_vect_params_ + i_vparam;
  }
  else {
    throw ngpu_exception(std::string("Unrecognized parameter ")
			 + param_name);
  }
}

float BaseNeuron::GetSpikeActivity(int i_neuron)
{
  int i_spike_buffer = i_neuron + i_node_0_;
  int Ns;
  gpuErrchk(hipMemcpy(&Ns, d_SpikeBufferSize + i_spike_buffer,
		       sizeof(int), hipMemcpyDeviceToHost));
  if (Ns==0) {
    return 0.0;
  }
  int time_idx;
  // get first (most recent) spike from buffer
  gpuErrchk(hipMemcpy(&time_idx, d_SpikeBufferTimeIdx + i_spike_buffer,
		       sizeof(int), hipMemcpyDeviceToHost));
  if (time_idx!=0) { // neuron is not spiking now
    return 0.0;
  }
  float spike_height;
  gpuErrchk(hipMemcpy(&spike_height, d_SpikeBufferHeight + i_spike_buffer,
		       sizeof(float), hipMemcpyDeviceToHost));

  return spike_height;
}

