#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include "spike_buffer.h"
#include "cuda_error.h"
#include "syn_model.h"

#define SPIKE_TIME_DIFF_GUARD 15000 // must be less than 16384
#define SPIKE_TIME_DIFF_THR 10000 // must be less than GUARD

extern __constant__ int NeuralGPUTimeIdx;
extern __constant__ float NeuralGPUTimeResolution;

unsigned int *d_RevSpikeNum;
unsigned int *d_RevSpikeTarget;
int *d_RevSpikeNConn;

extern __device__ void SynapseUpdate(int syn_group, float *w, float Dt);

__device__ unsigned int *RevSpikeNum;
__device__ unsigned int *RevSpikeTarget;
__device__ int *RevSpikeNConn;


//////////////////////////////////////////////////////////////////////
// This is the function called by the nested loop
// that makes use of positive post-pre spike time difference
__device__ void NestedLoopFunction1(int i_spike, int i_target_rev_conn)
{
  unsigned int target = RevSpikeTarget[i_spike];
  unsigned int i_conn = TargetRevConnection[target][i_target_rev_conn];
  unsigned char syn_group = ConnectionSynGroup[i_conn];
  if (syn_group>0) {
    float *weight = &ConnectionWeight[i_conn];
    int spike_time_idx = ConnectionSpikeTime[i_conn];
    int Dt_int = ((int)NeuralGPUTimeIdx - spike_time_idx)&0xffff;
    //if (Dt<0) { // there was no spike from this connection
    //  return;
    //}
    // The following lines are for solving the problem of limited size of
    // connection spike time
    //if (Dt>SPIKE_TIME_DIFF_THR) { // there was no spike from this connection
    //  return;
    //}
    //if (Dt==SPIKE_TIME_DIFF_THR) { // there was no spike from this connection
      // but due to the increase of time idx the difference
      // reached the threshold, so let's put it well above threshold
    //  ConnectionSpikeTime[i_conn]
    //	= (unsigned short)((NeuralGPUTimeIdx + SPIKE_TIME_DIFF_GUARD)&0xffff);
    //  return;
    //}
    if (Dt_int>=0 && Dt_int<MAX_SYN_DT) {
      SynapseUpdate(syn_group, weight, NeuralGPUTimeResolution*Dt_int);
    }
  }
}
	    

__global__ void RevSpikeBufferUpdate(unsigned int n_node)
{
  unsigned int i_node = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_node >= n_node) {
    return;
  }
  int target_spike_time_idx = LastSpikeTimeIdx[i_node];
  // Check if neuron is spiking now
  if (target_spike_time_idx!=NeuralGPUTimeIdx) {
    return;
  }
  int n_conn = TargetRevConnectionSize[i_node];
  if (n_conn>0) {
    unsigned int pos = atomicAdd(RevSpikeNum, 1);
    RevSpikeTarget[pos] = i_node;
    RevSpikeNConn[pos] = n_conn;
  }
}

__global__ void SetConnectionSpikeTime(unsigned int n_conn,
				       unsigned short time_idx)
{
  unsigned int i_conn = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_conn>=n_conn) {
    return;
  }
  ConnectionSpikeTime[i_conn] = time_idx;
}

__global__ void DeviceRevSpikeInit(unsigned int *rev_spike_num,
				   unsigned int *rev_spike_target,
				   int *rev_spike_n_conn)
{
  RevSpikeNum = rev_spike_num;
  RevSpikeTarget = rev_spike_target;
  RevSpikeNConn = rev_spike_n_conn;
  *RevSpikeNum = 0;
}

__global__ void RevSpikeReset()
{
  *RevSpikeNum = 0;
}
  

int RevSpikeInit(NetConnection *net_connection, int time_min_idx)
{
  int n_spike_buffers = net_connection->connection_.size();
  
  SetConnectionSpikeTime
    <<<(net_connection->StoredNConnections()+1023)/1024, 1024>>>
    (net_connection->StoredNConnections(),
     time_min_idx + SPIKE_TIME_DIFF_GUARD);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  gpuErrchk(hipMalloc(&d_RevSpikeNum, sizeof(unsigned int)));
  
  gpuErrchk(hipMalloc(&d_RevSpikeTarget,
		       n_spike_buffers*sizeof(unsigned int)));
  gpuErrchk(hipMalloc(&d_RevSpikeNConn,
		       n_spike_buffers*sizeof(int)));

  DeviceRevSpikeInit<<<1,1>>>(d_RevSpikeNum, d_RevSpikeTarget,
			      d_RevSpikeNConn);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  return 0;
}


int RevSpikeFree()
{
  gpuErrchk(hipFree(&d_RevSpikeNum));
  gpuErrchk(hipFree(&d_RevSpikeTarget));
  gpuErrchk(hipFree(&d_RevSpikeNConn));

  return 0;
}
