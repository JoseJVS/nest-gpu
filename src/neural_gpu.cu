#include "hip/hip_runtime.h"
/*
Copyright (C) 2016 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include <string>
#include <algorithm>
#include <mpi.h>
#include <hiprand.h>
//#include "connect.h"
#include "spike_buffer.h"
#include "rk5.h"
#include "cuda_error.h"
#include "aeif.h"
#include "send_spike.h"
#include "get_spike.h"
#include "connect_mpi.h"
#include "spike_mpi.h"
#include "spike_generator.h"
#include "multimeter.h"
#include "prefix_scan.h"
#include "getRealTime.h"
#include "random.h"
#include "neural_gpu.h"

#ifdef _OPENMP
#include <omp.h>
#define THREAD_MAXNUM omp_get_max_threads()
#define THREAD_IDX omp_get_thread_num()
#else
#define THREAD_MAXNUM 1
#define THREAD_IDX 0
#endif

using namespace std;

#define VERBOSE_TIME

NeuralGPU::NeuralGPU()
{
  random_generator_ = new hiprandGenerator_t;
  CURAND_CALL(hiprandCreateGenerator(random_generator_,
				    HIPRAND_RNG_PSEUDO_DEFAULT));
  poiss_generator_ = new PoissonGenerator;
  spike_generator_ = new SpikeGenerator;
  multimeter_ = new Multimeter;
  aeif_ = new AEIF;
  net_connection_ = new NetConnection;
  connect_mpi_ = new ConnectMpi;
  prefix_scan_ = new PrefixScan;

  SetRandomSeed(54321ULL);
  
  start_real_time_ = getRealTime();
  max_spike_buffer_num_ = 100;
  t_min_ = 0.0;
  sim_time_ = 1000.0;        //Simulation time in ms
  n_neurons_ = 0;
  n_poiss_nodes_ = 0;
  n_spike_gen_nodes_ = 0;
  SetTimeResolution(0.1);  // time resolution in ms
  /////ConnectMpiInit(&argc, &argv, time_resolution_);
  connect_mpi_->net_connection_ = net_connection_;
  prefix_scan_->Init();
}

NeuralGPU::~NeuralGPU()
{
  CURAND_CALL(hiprandDestroyGenerator(*random_generator_));
  delete poiss_generator_;
  delete spike_generator_;
  delete multimeter_;
  delete aeif_;
  delete net_connection_;
  delete connect_mpi_;
  delete prefix_scan_;
}

int NeuralGPU::SetRandomSeed(unsigned long long seed)
{
  CURAND_CALL(hiprandDestroyGenerator(*random_generator_));
  random_generator_ = new hiprandGenerator_t;
  CURAND_CALL(hiprandCreateGenerator(random_generator_,
				    HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(*random_generator_, seed));
  poiss_generator_->random_generator_ = random_generator_;

  return 0;
}

int NeuralGPU::SetTimeResolution(float time_res)
{
  time_resolution_ = time_res;
  net_connection_->time_resolution_ = time_res;
  
  return 0;
}

int NeuralGPU::CreateNeuron(int n_neurons, int n_receptors)
{
  if (n_neurons_ != 0) {
    cerr << "Number of neurons cannot be modified.\n";
    exit(0);
  }
  else if (n_neurons <= 0) {
    cerr << "Number of neurons must be greater than zero.\n";
    exit(0);
  }
  else if (n_receptors <= 0) {
    cerr << "Number of receptors must be greater than zero.\n";
    exit(0);
  }

  n_neurons_ = n_neurons;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_neurons_, conn);

  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_neurons_, conn_node);

  //SpikeInit(max_spike_num_);
  aeif_->Init(i_node_0, n_neurons_, n_receptors);
  
  return i_node_0;
}

int NeuralGPU::CreatePoissonGenerator(int n_nodes, float rate)
{
  if (n_poiss_nodes_ != 0) {
    cerr << "Number of poisson generators cannot be modified.\n";
    exit(0);
  }
  else if (n_nodes <= 0) {
    cerr << "Number of nodes must be greater than zero.\n";
    exit(0);
  }

  n_poiss_nodes_ = n_nodes;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_poiss_nodes_, conn);

  if ((int)connect_mpi_->extern_connection_.size() != i_node_0) {
    cerr << "Error: net_connection_.connection_ and "
      "connect_mpi_.extern_connection_ must have the same size!\n";
  }
  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_poiss_nodes_, conn_node);

  float lambda = rate*time_resolution_ / 1000.0; // rate is in Hz, time in ms
  poiss_generator_->Create(random_generator_, i_node_0, n_poiss_nodes_, lambda);
  
  return i_node_0;
}

int NeuralGPU::CreateSpikeGenerator(int n_nodes)
{
  if (n_spike_gen_nodes_ != 0) {
    cerr << "Number of spike generators cannot be modified.\n";
    exit(0);
  }
  else if (n_nodes <= 0) {
    cerr << "Number of nodes must be greater than zero.\n";
    exit(0);
  }

  n_spike_gen_nodes_ = n_nodes;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_spike_gen_nodes_, conn);

  if ((int)connect_mpi_->extern_connection_.size() != i_node_0) {
    cerr << "Error: net_connection_.connection_ and "
      "connect_mpi_.extern_connection_ must have the same size!\n";
  }
  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_spike_gen_nodes_, conn_node);

  spike_generator_->Create(i_node_0, n_spike_gen_nodes_,
			  t_min_, time_resolution_);
  
  return i_node_0;
}

int NeuralGPU::Simulate()
{
  double SpikeBufferUpdate_time = 0;
  double poisson_generator_time = 0;
  double spike_generator_time = 0;
  double aeif_Update_time = 0;
  double copy_ext_spike_time = 0;
  double SendExternalSpike_time = 0;
  double SendSpikeToRemote_time = 0;
  double RecvSpikeFromRemote_time = 0;
  double PrefixScan_time = 0;
  double GetSpike_time = 0;
  double SpikeReset_time = 0;
  double ExternalSpikeReset_time = 0;
  double time_mark;
  
  float t_min = 0.0;

  max_spike_num_ = net_connection_->connection_.size();
  max_spike_per_host_ = net_connection_->connection_.size();

  SpikeInit(max_spike_num_);
  SpikeBufferInit(net_connection_, max_spike_buffer_num_);

  // remove superfluous argument mpi_np
  connect_mpi_->ExternalSpikeInit(connect_mpi_->extern_connection_.size(),
				 max_spike_num_, connect_mpi_->mpi_np_,
				 max_spike_per_host_);
  InitGetSpikeArray(n_neurons_, aeif_->n_receptors_);

  //////////////////////////////////////////////////
  //char filename[100];
  //sprintf(filename, "test_arr_%d.dat", connect_mpi_.mpi_id_);
  //FILE *fp=fopen(filename, "wb");
  multimeter_->OpenFiles();
  
  int Nt=(int)round(sim_time_/time_resolution_);
  printf("%d\n", Nt);

  aeif_->Calibrate(t_min);

  //float x;
  //float y;
  //aeif_.GetX(test_arr_idx, 1, &x);
  //aeif_.GetY(test_var_idx, test_arr_idx, 1, &y);
  //fprintf(fp,"%f\t%f\n", x, y);

///////////////////////////////////
  multimeter_->WriteRecords();
  
  build_real_time_ = getRealTime();
  
  cout << "Simulating on host " << connect_mpi_->mpi_id_ << " ..." <<endl;

  for (int it=0; it<Nt; it++) {
    float t1 = t_min_ + time_resolution_*(it + 1);
    if (it%100==0)
      printf("%d\n", it);

    time_mark = getRealTime();
    SpikeBufferUpdate<<<(net_connection_->connection_.size()+1023)/1024,
      1024>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    SpikeBufferUpdate_time += (getRealTime() - time_mark);

    time_mark = getRealTime();
    if (n_poiss_nodes_>0) {
      poiss_generator_->Update(Nt-it);
      poisson_generator_time += (getRealTime() - time_mark);
    }

    time_mark = getRealTime();
    if (n_spike_gen_nodes_>0) {
      spike_generator_->Update(it);
      spike_generator_time += (getRealTime() - time_mark);
    }

    time_mark = getRealTime();
    aeif_->Update(it, t1);
    aeif_Update_time += (getRealTime() - time_mark);
    
    multimeter_->WriteRecords();
    
    int n_ext_spike;
    time_mark = getRealTime();
    gpuErrchk(hipMemcpy(&n_ext_spike, d_ExternalSpikeNum, sizeof(int),
			 hipMemcpyDeviceToHost));
    copy_ext_spike_time += (getRealTime() - time_mark);
    
    if (n_ext_spike != 0) {
      //cout << "n_ext_spike " << n_ext_spike << endl;
      time_mark = getRealTime();
      SendExternalSpike<<<(n_ext_spike+1023)/1024, 1024>>>();
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      SendExternalSpike_time += (getRealTime() - time_mark);
      
    }
    for (int ih=0; ih<connect_mpi_->mpi_np_; ih++) {

      if (ih == connect_mpi_->mpi_id_) {
	time_mark = getRealTime();
	connect_mpi_->SendSpikeToRemote(connect_mpi_->mpi_np_,
				       max_spike_per_host_);
	SendSpikeToRemote_time += (getRealTime() - time_mark);
      }
      else {
	time_mark = getRealTime();
	connect_mpi_->RecvSpikeFromRemote(ih, max_spike_per_host_);
	RecvSpikeFromRemote_time += (getRealTime() - time_mark);
      }
    }

    int n_spikes;
    time_mark = getRealTime();
    gpuErrchk(hipMemcpy(&n_spikes, d_SpikeNum, sizeof(int),
			 hipMemcpyDeviceToHost));
    //cout << "n_spikes: " << n_spikes << endl;
    if (n_spikes > 0) {
      prefix_scan_->Scan(d_SpikeTargetNumSum, d_SpikeTargetNum, n_spikes);
      uint n_get_spikes;
      gpuErrchk(hipMemcpy(&n_get_spikes, &d_SpikeTargetNumSum[n_spikes],
			   sizeof(uint), hipMemcpyDeviceToHost));
      PrefixScan_time += (getRealTime() - time_mark);
      if(n_get_spikes>0) {
        ClearGetSpikeArray(n_neurons_, aeif_->n_receptors_);

	time_mark = getRealTime();
	uint grid_dim_x, grid_dim_y;
	if (n_get_spikes<65536*1024) { // max grid dim * max block dim
	  grid_dim_x = (n_get_spikes+1023)/1024;
	  grid_dim_y = 1;
	}
	else {
	  grid_dim_x = 64; // I think it's not necessary to increase it
	  if (n_get_spikes>grid_dim_x*1024*65535) {
	    cerr << "n_get_spikes " << n_get_spikes << " larger than threshold "
		 << grid_dim_x*1024*65535 << " .\n";
	    exit(-1);
	  }
	  grid_dim_y = (n_get_spikes + grid_dim_x*1024 -1) / (grid_dim_x*1024);
	}
	dim3 numBlocks(grid_dim_x, grid_dim_y);
	CollectSpikes<<<numBlocks, 1024>>>(n_get_spikes, aeif_->n_var_,
				      aeif_->n_params_);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

        // improve using a grid
        GetSpikes<<<(n_neurons_*aeif_->n_receptors_+1023)/1024, 1024>>>
                    (aeif_->n_receptors_);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	GetSpike_time += (getRealTime() - time_mark);
      }
    }
    time_mark = getRealTime();
    SpikeReset<<<1, 1>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    SpikeReset_time += (getRealTime() - time_mark);
    
    time_mark = getRealTime();
    ExternalSpikeReset<<<1, 1>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    ExternalSpikeReset_time += (getRealTime() - time_mark);
  }
  end_real_time_ = getRealTime();
  //fclose(fp);
  multimeter_->CloseFiles();
  //aeif.rk5.Free();

#ifdef VERBOSE_TIME
  cout << endl;
  cout << "  SpikeBufferUpdate_time: " << SpikeBufferUpdate_time << endl;
  cout << "  poisson_generator_time: " << poisson_generator_time << endl;
  cout << "  spike_generator_time: " << spike_generator_time << endl;
  cout << "  aeif_Update_time: " << aeif_Update_time << endl;
  cout << "  copy_ext_spike_time: " << copy_ext_spike_time << endl;
  cout << "  SendExternalSpike_time: " << SendExternalSpike_time << endl;
  cout << "  SendSpikeToRemote_time: " << SendSpikeToRemote_time << endl;
  cout << "  RecvSpikeFromRemote_time: " << RecvSpikeFromRemote_time << endl;
  cout << "  PrefixScan_time: " << PrefixScan_time << endl;
  cout << "  GetSpike_time: " << GetSpike_time << endl;
  cout << "  SpikeReset_time: " << SpikeReset_time << endl;
  cout << "  ExternalSpikeReset_time: " << ExternalSpikeReset_time << endl;
#endif
  printf("Build real time = %lf\n",
	 (build_real_time_ - start_real_time_));
  printf("Simulation real time = %lf\n",
	 (end_real_time_ - build_real_time_));

  return 0;
}

int NeuralGPU::CreateRecord(string file_name, string var_name, int *i_neurons,
			    int n_neurons)
{
  return multimeter_->CreateRecord(aeif_, file_name, var_name, i_neurons,
				  n_neurons);
}

int NeuralGPU::ConnectFixedIndegree
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay, int indegree
 )
{
  unsigned int *rnd = RandomInt(n_target_neurons*indegree);
  vector<int> input_array;
  for (int i=0; i<n_source_neurons; i++) {
    input_array.push_back(i_source_neuron_0 + i);
  }
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared) collapse(2)
#endif
  for (int k=0; k<n_target_neurons; k++) {
    for (int i=0; i<indegree; i++) {
      int j = i + rnd[k*indegree+i] % (n_source_neurons - i);
#ifdef _OPENMP
      omp_set_lock(&(lock[i]));
#endif
      if (j!=i) {
#ifdef _OPENMP
	omp_set_lock(&(lock[j]));
#endif
	swap(input_array[i], input_array[j]);
#ifdef _OPENMP
	omp_unset_lock(&(lock[j]));
#endif
      }
      int itn = k + i_target_neuron_0;
      int isn = input_array[i];
      net_connection_->Connect(isn, itn, i_port, weight, delay);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
    }
  }
  delete[] rnd;
#ifdef _OPENMP
  delete[] lock;
#endif
  
  return 0;
}

int NeuralGPU::ConnectAllToAll
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay
 )
{
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared) collapse(2)
#endif
  for (int itn=i_target_neuron_0; itn<i_target_neuron_0+n_target_neurons;
       itn++) {
    for (int i=0; i<n_source_neurons; i++) {
      int isn = i_source_neuron_0 + i;
#ifdef _OPENMP
      omp_set_lock(&(lock[i]));
#endif
      net_connection_->Connect(isn, itn, i_port, weight, delay);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
    }
  }

#ifdef _OPENMP
  delete[] lock;
#endif

  return 0;
}

int NeuralGPU::ConnectOneToOne
(
 int i_source_neuron_0, int i_target_neuron_0, int n_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  for (int in=0; in<n_neurons; in++) {
    net_connection_->Connect(i_source_neuron_0+in,i_target_neuron_0+in ,
			    i_port, weight, delay);
  }

  return 0;
}

int NeuralGPU::SetNeuronParams(string param_name, int i_node, int n_neurons,
			       float val)
{
  int i_neuron = i_node - aeif_->i_node_0_;
  
  return aeif_->SetParams(param_name, i_neuron, n_neurons, val);
}

int NeuralGPU::SetNeuronVectParams(string param_name, int i_node, int n_neurons,
				   float *params, int vect_size)
{
  int i_neuron = i_node - aeif_->i_node_0_;
  
  return aeif_->SetVectParams(param_name, i_neuron, n_neurons, params,
			     vect_size);
}

int NeuralGPU::ConnectMpiInit(int argc, char *argv[])
{
  return connect_mpi_->MpiInit(argc, argv);
}

int NeuralGPU::MpiId()
{
  return connect_mpi_->mpi_id_;
}

int NeuralGPU::MpiNp()
{
  return connect_mpi_->mpi_np_;
}

int NeuralGPU::ProcMaster()
{
  return connect_mpi_->ProcMaster();
}

int NeuralGPU::MpiFinalize()
{
  return MPI_Finalize();
}

int NeuralGPU::SetSpikeGenerator(int i_node, int n_spikes, float *spike_time,
				 float *spike_height)
{
  return spike_generator_->Set(i_node, n_spikes, spike_time, spike_height);
}

unsigned int *NeuralGPU::RandomInt(size_t n)
{
  return curand_int(*random_generator_, n);
}


int NeuralGPU::RemoteConnectFixedIndegree
(
 int i_source_host, int i_source_neuron_0, int n_source_neurons,
 int i_target_host, int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay, int indegree
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectFixedIndegree(i_source_neuron_0, n_source_neurons, i_target_neuron_0,
			 n_target_neurons, i_port, weight, delay, indegree);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_target_neurons*indegree];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_target_neurons*indegree, i_source_host);

      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<indegree; i++) {
      	  int i_remote_neuron = i_remote_neuron_arr[k*indegree+i];
	  int i_target_neuron = k + i_target_neuron_0;
	  net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
	}
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      unsigned int *rnd = RandomInt(n_target_neurons*indegree); // check parall. seed problem
      vector<int> input_array;
      for (int i=0; i<n_source_neurons; i++) {
	input_array.push_back(i_source_neuron_0 + i);
      }
      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<indegree; i++) {
	  int j = i + rnd[k*indegree+i] % (n_source_neurons - i);
	  if (j!=i) {
	    swap(input_array[i], input_array[j]);
	  }
	  int i_source_neuron = input_array[i];
	  
	  int i_remote_neuron = -1;
	  for (vector<ExternalConnectionNode >::iterator it =
		 connect_mpi_->extern_connection_[i_source_neuron].begin();
	       it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	    if ((*it).target_host_id == i_target_host) {
	      i_remote_neuron = (*it).remote_neuron_id;
	      break;
	    }
	  }
	  if (i_remote_neuron == -1) {
	    i_remote_neuron = i_new_remote_neuron;
	    i_new_remote_neuron++;
	    ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	    connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	  }
	  i_remote_neuron_arr[k*indegree+i] = i_remote_neuron;
	}
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_target_neurons*indegree, i_target_host);
      delete[] rnd;
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnectAllToAll
(
 int i_source_host, int i_source_neuron_0, int n_source_neurons,
 int i_target_host, int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectAllToAll(i_source_neuron_0, n_source_neurons, i_target_neuron_0,
			 n_target_neurons, i_port, weight, delay);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_target_neurons*n_source_neurons];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_target_neurons*n_source_neurons,
				 i_source_host);

      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<n_source_neurons; i++) {
      	  int i_remote_neuron = i_remote_neuron_arr[k*n_source_neurons+i];
	  int i_target_neuron = k + i_target_neuron_0;
	  net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
	}
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<n_source_neurons; i++) {
	  int i_source_neuron = i + i_source_neuron_0;
	  
	  int i_remote_neuron = -1;
	  for (vector<ExternalConnectionNode >::iterator it =
		 connect_mpi_->extern_connection_[i_source_neuron].begin();
	       it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	    if ((*it).target_host_id == i_target_host) {
	      i_remote_neuron = (*it).remote_neuron_id;
	      break;
	    }
	  }
	  if (i_remote_neuron == -1) {
	    i_remote_neuron = i_new_remote_neuron;
	    i_new_remote_neuron++;
	    ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	    connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	  }
	  i_remote_neuron_arr[k*n_source_neurons+i] = i_remote_neuron;
	}
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_target_neurons*n_source_neurons,
				 i_target_host);
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnectOneToOne
(
 int i_source_host, int i_source_neuron_0,
 int i_target_host, int i_target_neuron_0, int n_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectOneToOne(i_source_neuron_0, i_target_neuron_0,
			 n_neurons, i_port, weight, delay);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_neurons];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_neurons, i_source_host);

      for (int i=0; i<n_neurons; i++) {
	int i_remote_neuron = i_remote_neuron_arr[i];
	int i_target_neuron = i + i_target_neuron_0;
	net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      for (int i=0; i<n_neurons; i++) {
	int i_source_neuron = i + i_source_neuron_0;
	  
	int i_remote_neuron = -1;
	for (vector<ExternalConnectionNode >::iterator it =
	       connect_mpi_->extern_connection_[i_source_neuron].begin();
	     it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	  if ((*it).target_host_id == i_target_host) {
	    i_remote_neuron = (*it).remote_neuron_id;
	    break;
	  }
	}
	if (i_remote_neuron == -1) {
	  i_remote_neuron = i_new_remote_neuron;
	  i_new_remote_neuron++;
	  ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	  connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	}
	i_remote_neuron_arr[i] = i_remote_neuron;
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_neurons, i_target_host);
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnect(int i_source_host, int i_source_neuron,
			     int i_target_host, int i_target_neuron,
			     unsigned char i_port, float weight, float delay)
{
  return connect_mpi_->RemoteConnect(i_source_host, i_source_neuron,
				     i_target_host, i_target_neuron,
				     i_port, weight, delay);
}

