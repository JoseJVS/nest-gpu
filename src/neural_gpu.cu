#include "hip/hip_runtime.h"
/*
Copyright (C) 2019 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include <string>
#include <algorithm>
#include <mpi.h>
#include <hiprand.h>
//#include "connect.h"
#include "spike_buffer.h"
#include "cuda_error.h"
#include "aeif.h"
#include "send_spike.h"
#include "get_spike.h"
#include "connect_mpi.h"
#include "spike_mpi.h"
#include "spike_generator.h"
#include "multimeter.h"
#include "getRealTime.h"
#include "random.h"
#include "neural_gpu.h"
#include "nested_loop.h"

#ifdef _OPENMP
#include <omp.h>
#define THREAD_MAXNUM omp_get_max_threads()
#define THREAD_IDX omp_get_thread_num()
#else
#define THREAD_MAXNUM 1
#define THREAD_IDX 0
#endif

using namespace std;

#define VERBOSE_TIME

NeuralGPU::NeuralGPU()
{
  random_generator_ = new hiprandGenerator_t;
  CURAND_CALL(hiprandCreateGenerator(random_generator_,
				    HIPRAND_RNG_PSEUDO_DEFAULT));
  poiss_generator_ = new PoissonGenerator;
  spike_generator_ = new SpikeGenerator;
  multimeter_ = new Multimeter;
  AEIF *aeif_neuron = new AEIF;
  neuron_vect_.push_back(aeif_neuron);
  net_connection_ = new NetConnection;
  connect_mpi_ = new ConnectMpi;

  SetRandomSeed(54321ULL);
  
  start_real_time_ = getRealTime();
  max_spike_buffer_num_ = 100;
  t_min_ = 0.0;
  sim_time_ = 1000.0;        //Simulation time in ms
  n_neurons_ = 0;
  n_poiss_nodes_ = 0;
  n_spike_gen_nodes_ = 0;
  SetTimeResolution(0.1);  // time resolution in ms
  /////ConnectMpiInit(&argc, &argv, time_resolution_);
  connect_mpi_->net_connection_ = net_connection_;
  NestedLoop::Init();
}

NeuralGPU::~NeuralGPU()
{
  CURAND_CALL(hiprandDestroyGenerator(*random_generator_));
  delete poiss_generator_;
  delete spike_generator_;
  delete multimeter_;
  delete neuron_vect_[0];
  delete net_connection_;
  delete connect_mpi_;
  FreeNeuronGroupMap();
  FreeGetSpikeArrays();
}

int NeuralGPU::SetRandomSeed(unsigned long long seed)
{
  CURAND_CALL(hiprandDestroyGenerator(*random_generator_));
  random_generator_ = new hiprandGenerator_t;
  CURAND_CALL(hiprandCreateGenerator(random_generator_,
				    HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(*random_generator_, seed));
  poiss_generator_->random_generator_ = random_generator_;

  return 0;
}

int NeuralGPU::SetTimeResolution(float time_res)
{
  time_resolution_ = time_res;
  net_connection_->time_resolution_ = time_res;
  
  return 0;
}

int NeuralGPU::CreateNeuron(int n_neurons, int n_receptors)
{
  if (n_neurons_ != 0) {
    cerr << "Number of neurons cannot be modified.\n";
    exit(0);
  }
  else if (n_neurons <= 0) {
    cerr << "Number of neurons must be greater than zero.\n";
    exit(0);
  }
  else if (n_receptors <= 0) {
    cerr << "Number of receptors must be greater than zero.\n";
    exit(0);
  }

  n_neurons_ = n_neurons;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_neurons, conn);

  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_neurons, conn_node);

  int i_neuron_group = InsertNeuronGroup(n_neurons, n_receptors);
  neuron_vect_[0]->Init(i_node_0, n_neurons, n_receptors, i_neuron_group);
  
  return i_node_0;
}

int NeuralGPU::CreatePoissonGenerator(int n_nodes, float rate)
{
  if (n_poiss_nodes_ != 0) {
    cerr << "Number of poisson generators cannot be modified.\n";
    exit(0);
  }
  else if (n_nodes <= 0) {
    cerr << "Number of nodes must be greater than zero.\n";
    exit(0);
  }

  n_poiss_nodes_ = n_nodes;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_poiss_nodes_, conn);

  if ((int)connect_mpi_->extern_connection_.size() != i_node_0) {
    cerr << "Error: net_connection_.connection_ and "
      "connect_mpi_.extern_connection_ must have the same size!\n";
  }
  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_poiss_nodes_, conn_node);

  float lambda = rate*time_resolution_ / 1000.0; // rate is in Hz, time in ms
  poiss_generator_->Create(random_generator_, i_node_0, n_poiss_nodes_, lambda);
  InsertNeuronGroup(n_nodes, 0);
    
  return i_node_0;
}

int NeuralGPU::CreateSpikeGenerator(int n_nodes)
{
  if (n_spike_gen_nodes_ != 0) {
    cerr << "Number of spike generators cannot be modified.\n";
    exit(0);
  }
  else if (n_nodes <= 0) {
    cerr << "Number of nodes must be greater than zero.\n";
    exit(0);
  }

  n_spike_gen_nodes_ = n_nodes;               

  int i_node_0 = net_connection_->connection_.size();
  
  vector<ConnGroup> conn;
  vector<vector<ConnGroup> >:: iterator it
    = net_connection_->connection_.end();
  net_connection_->connection_.insert(it, n_spike_gen_nodes_, conn);

  if ((int)connect_mpi_->extern_connection_.size() != i_node_0) {
    cerr << "Error: net_connection_.connection_ and "
      "connect_mpi_.extern_connection_ must have the same size!\n";
  }
  vector<ExternalConnectionNode > conn_node;
  vector<vector< ExternalConnectionNode> >::iterator it1
    = connect_mpi_->extern_connection_.end();
  connect_mpi_->extern_connection_.insert(it1, n_spike_gen_nodes_, conn_node);

  spike_generator_->Create(i_node_0, n_spike_gen_nodes_,
			  t_min_, time_resolution_);
  InsertNeuronGroup(n_nodes, 0);
  
  return i_node_0;
}

int NeuralGPU::Simulate()
{
  double SpikeBufferUpdate_time = 0;
  double poisson_generator_time = 0;
  double spike_generator_time = 0;
  double neuron_Update_time = 0;
  double copy_ext_spike_time = 0;
  double SendExternalSpike_time = 0;
  double SendSpikeToRemote_time = 0;
  double RecvSpikeFromRemote_time = 0;
  double NestedLoop_time = 0;
  double GetSpike_time = 0;
  double SpikeReset_time = 0;
  double ExternalSpikeReset_time = 0;
  double time_mark;
  
  float t_min = 0.0;

  NeuronGroupArrayInit();
  
  max_spike_num_ = net_connection_->connection_.size()
    * net_connection_->MaxDelayNum();
  
  max_spike_per_host_ = net_connection_->connection_.size()
    * net_connection_->MaxDelayNum();

  SpikeInit(max_spike_num_);
  SpikeBufferInit(net_connection_, max_spike_buffer_num_);

  // remove superfluous argument mpi_np
  connect_mpi_->ExternalSpikeInit(connect_mpi_->extern_connection_.size(),
				 max_spike_num_, connect_mpi_->mpi_np_,
				 max_spike_per_host_);

  multimeter_->OpenFiles();
  
  int Nt=(int)round(sim_time_/time_resolution_);
  printf("%d\n", Nt);

  neuron_vect_[0]->Calibrate(t_min);

  //float x;
  //float y;
  //neuron_vect_[0].GetX(test_arr_idx, 1, &x);
  //neuron_vect_[0].GetY(test_var_idx, test_arr_idx, 1, &y);
  //fprintf(fp,"%f\t%f\n", x, y);

///////////////////////////////////
  multimeter_->WriteRecords();
  
  build_real_time_ = getRealTime();
  
  cout << "Simulating on host " << connect_mpi_->mpi_id_ << " ..." <<endl;

  for (int it=0; it<Nt; it++) {
    float t1 = t_min_ + time_resolution_*(it + 1);
    if (it%100==0)
      printf("%d\n", it);

    time_mark = getRealTime();
    SpikeBufferUpdate<<<(net_connection_->connection_.size()+1023)/1024,
      1024>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    SpikeBufferUpdate_time += (getRealTime() - time_mark);
    time_mark = getRealTime();
    if (n_poiss_nodes_>0) {
      poiss_generator_->Update(Nt-it);
      poisson_generator_time += (getRealTime() - time_mark);
    }

    time_mark = getRealTime();
    if (n_spike_gen_nodes_>0) {
      spike_generator_->Update(it);
      spike_generator_time += (getRealTime() - time_mark);
    }

    time_mark = getRealTime();
    neuron_vect_[0]->Update(it, t1);
    neuron_Update_time += (getRealTime() - time_mark);
    multimeter_->WriteRecords();
    int n_ext_spike;
    time_mark = getRealTime();
    gpuErrchk(hipMemcpy(&n_ext_spike, d_ExternalSpikeNum, sizeof(int),
			 hipMemcpyDeviceToHost));
    copy_ext_spike_time += (getRealTime() - time_mark);

    if (n_ext_spike != 0) {
      //cout << "n_ext_spike " << n_ext_spike << endl;
      time_mark = getRealTime();
      SendExternalSpike<<<(n_ext_spike+1023)/1024, 1024>>>();
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      SendExternalSpike_time += (getRealTime() - time_mark);
      
    }
    for (int ih=0; ih<connect_mpi_->mpi_np_; ih++) {

      if (ih == connect_mpi_->mpi_id_) {
	time_mark = getRealTime();
	connect_mpi_->SendSpikeToRemote(connect_mpi_->mpi_np_,
				       max_spike_per_host_);
	SendSpikeToRemote_time += (getRealTime() - time_mark);
      }
      else {
	time_mark = getRealTime();
	connect_mpi_->RecvSpikeFromRemote(ih, max_spike_per_host_);
	RecvSpikeFromRemote_time += (getRealTime() - time_mark);
      }
    }

    int n_spikes;
    time_mark = getRealTime();
    gpuErrchk(hipMemcpy(&n_spikes, d_SpikeNum, sizeof(int),
			 hipMemcpyDeviceToHost));
    //cout << "n_spikes: " << n_spikes << endl;
    if (n_spikes > 0) {
      ClearGetSpikeArrays();      
      time_mark = getRealTime();
      NestedLoop::Run(n_spikes, d_SpikeTargetNum);

      //, neuron_vect_[0]->n_var_,
      // neuron_vect_[0]->n_params_);
      NestedLoop_time += (getRealTime() - time_mark);
      time_mark = getRealTime();
      // improve using a grid
      GetSpikes<<<(neuron_vect_[0]->n_neurons_
		   *neuron_vect_[0]->n_receptors_+1023)/1024, 1024>>>
	(neuron_vect_[0]->i_neuron_group_, neuron_vect_[0]->n_neurons_,
	 neuron_vect_[0]->n_receptors_,
	 neuron_vect_[0]->n_var_,
	 neuron_vect_[0]->receptor_weight_arr_,
	 neuron_vect_[0]->receptor_weight_arr_step_,
	 neuron_vect_[0]->receptor_weight_port_step_,
	 //neuron_vect_[0]->GetVarArr());
	 neuron_vect_[0]->receptor_input_arr_,
	 neuron_vect_[0]->receptor_input_arr_step_,
	 neuron_vect_[0]->receptor_input_port_step_);

      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );

      GetSpike_time += (getRealTime() - time_mark);
    }
    time_mark = getRealTime();
    SpikeReset<<<1, 1>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    SpikeReset_time += (getRealTime() - time_mark);
    
    time_mark = getRealTime();
    ExternalSpikeReset<<<1, 1>>>();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    ExternalSpikeReset_time += (getRealTime() - time_mark);
  }
  end_real_time_ = getRealTime();

  multimeter_->CloseFiles();
  //neuron.rk5.Free();

#ifdef VERBOSE_TIME
  cout << endl;
  cout << "  SpikeBufferUpdate_time: " << SpikeBufferUpdate_time << endl;
  cout << "  poisson_generator_time: " << poisson_generator_time << endl;
  cout << "  spike_generator_time: " << spike_generator_time << endl;
  cout << "  neuron_Update_time: " << neuron_Update_time << endl;
  cout << "  copy_ext_spike_time: " << copy_ext_spike_time << endl;
  cout << "  SendExternalSpike_time: " << SendExternalSpike_time << endl;
  cout << "  SendSpikeToRemote_time: " << SendSpikeToRemote_time << endl;
  cout << "  RecvSpikeFromRemote_time: " << RecvSpikeFromRemote_time << endl;
  cout << "  NestedLoop_time: " << NestedLoop_time << endl;
  cout << "  GetSpike_time: " << GetSpike_time << endl;
  cout << "  SpikeReset_time: " << SpikeReset_time << endl;
  cout << "  ExternalSpikeReset_time: " << ExternalSpikeReset_time << endl;
#endif
  printf("Build real time = %lf\n",
	 (build_real_time_ - start_real_time_));
  printf("Simulation real time = %lf\n",
	 (end_real_time_ - build_real_time_));

  return 0;
}

int NeuralGPU::CreateRecord(string file_name, string var_name, int *i_neurons,
			    int n_neurons)
{
  return multimeter_->CreateRecord(neuron_vect_[0], file_name, var_name, i_neurons,
				  n_neurons);
}

int NeuralGPU::ConnectFixedIndegree
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay, int indegree
 )
{
  unsigned int *rnd = RandomInt(n_target_neurons*indegree);
  vector<int> input_array;
  for (int i=0; i<n_source_neurons; i++) {
    input_array.push_back(i_source_neuron_0 + i);
  }
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared) collapse(2)
#endif
  for (int k=0; k<n_target_neurons; k++) {
    for (int i=0; i<indegree; i++) {
      int j = i + rnd[k*indegree+i] % (n_source_neurons - i);
#ifdef _OPENMP
      omp_set_lock(&(lock[i]));
#endif
      if (j!=i) {
#ifdef _OPENMP
	omp_set_lock(&(lock[j]));
#endif
	swap(input_array[i], input_array[j]);
#ifdef _OPENMP
	omp_unset_lock(&(lock[j]));
#endif
      }
      int itn = k + i_target_neuron_0;
      int isn = input_array[i];
      net_connection_->Connect(isn, itn, i_port, weight, delay);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
    }
  }
  delete[] rnd;
#ifdef _OPENMP
  delete[] lock;
#endif
  
  return 0;
}

int NeuralGPU::ConnectAllToAll
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay
 )
{
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared) collapse(2)
#endif
  for (int itn=i_target_neuron_0; itn<i_target_neuron_0+n_target_neurons;
       itn++) {
    for (int i=0; i<n_source_neurons; i++) {
      int isn = i_source_neuron_0 + i;
#ifdef _OPENMP
      omp_set_lock(&(lock[i]));
#endif
      net_connection_->Connect(isn, itn, i_port, weight, delay);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
    }
  }

#ifdef _OPENMP
  delete[] lock;
#endif

  return 0;
}

int NeuralGPU::ConnectOneToOne
(
 int i_source_neuron_0, int i_target_neuron_0, int n_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  for (int in=0; in<n_neurons; in++) {
    net_connection_->Connect(i_source_neuron_0+in,i_target_neuron_0+in ,
			    i_port, weight, delay);
  }

  return 0;
}

int NeuralGPU::SetNeuronParams(string param_name, int i_node, int n_neurons,
			       float val)
{
  int i_neuron = i_node - neuron_vect_[0]->i_node_0_;
  
  return neuron_vect_[0]->SetScalParams(param_name, i_neuron, n_neurons, val);
}

int NeuralGPU::SetNeuronVectParams(string param_name, int i_node, int n_neurons,
				   float *params, int vect_size)
{
  int i_neuron = i_node - neuron_vect_[0]->i_node_0_;
  
  return neuron_vect_[0]->SetVectParams(param_name, i_neuron, n_neurons, params,
			     vect_size);
}

int NeuralGPU::ConnectMpiInit(int argc, char *argv[])
{
  return connect_mpi_->MpiInit(argc, argv);
}

int NeuralGPU::MpiId()
{
  return connect_mpi_->mpi_id_;
}

int NeuralGPU::MpiNp()
{
  return connect_mpi_->mpi_np_;
}

int NeuralGPU::ProcMaster()
{
  return connect_mpi_->ProcMaster();
}

int NeuralGPU::MpiFinalize()
{
  return MPI_Finalize();
}

int NeuralGPU::SetSpikeGenerator(int i_node, int n_spikes, float *spike_time,
				 float *spike_height)
{
  return spike_generator_->Set(i_node, n_spikes, spike_time, spike_height);
}

int NeuralGPU::RemoteConnectFixedIndegree
(
 int i_source_host, int i_source_neuron_0, int n_source_neurons,
 int i_target_host, int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay, int indegree
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectFixedIndegree(i_source_neuron_0, n_source_neurons, i_target_neuron_0,
			 n_target_neurons, i_port, weight, delay, indegree);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_target_neurons*indegree];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
      
      //NEW, CHECK ///////////
      InsertNeuronGroup(i_new_remote_neuron
			- net_connection_->connection_.size(), 0);
      ///////////////////////
      
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_target_neurons*indegree, i_source_host);

      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<indegree; i++) {
      	  int i_remote_neuron = i_remote_neuron_arr[k*indegree+i];
	  int i_target_neuron = k + i_target_neuron_0;
	  net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
	}
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      unsigned int *rnd = RandomInt(n_target_neurons*indegree); // check parall. seed problem
      vector<int> input_array;
      for (int i=0; i<n_source_neurons; i++) {
	input_array.push_back(i_source_neuron_0 + i);
      }
      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<indegree; i++) {
	  int j = i + rnd[k*indegree+i] % (n_source_neurons - i);
	  if (j!=i) {
	    swap(input_array[i], input_array[j]);
	  }
	  int i_source_neuron = input_array[i];
	  
	  int i_remote_neuron = -1;
	  for (vector<ExternalConnectionNode >::iterator it =
		 connect_mpi_->extern_connection_[i_source_neuron].begin();
	       it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	    if ((*it).target_host_id == i_target_host) {
	      i_remote_neuron = (*it).remote_neuron_id;
	      break;
	    }
	  }
	  if (i_remote_neuron == -1) {
	    i_remote_neuron = i_new_remote_neuron;
	    i_new_remote_neuron++;
	    ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	    connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	  }
	  i_remote_neuron_arr[k*indegree+i] = i_remote_neuron;
	}
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_target_neurons*indegree, i_target_host);
      delete[] rnd;
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnectAllToAll
(
 int i_source_host, int i_source_neuron_0, int n_source_neurons,
 int i_target_host, int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectAllToAll(i_source_neuron_0, n_source_neurons, i_target_neuron_0,
			 n_target_neurons, i_port, weight, delay);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_target_neurons*n_source_neurons];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
            
      //NEW, CHECK ///////////
      InsertNeuronGroup(i_new_remote_neuron
			- net_connection_->connection_.size(), 0);
      ///////////////////////
      
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_target_neurons*n_source_neurons,
				 i_source_host);

      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<n_source_neurons; i++) {
      	  int i_remote_neuron = i_remote_neuron_arr[k*n_source_neurons+i];
	  int i_target_neuron = k + i_target_neuron_0;
	  net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
	}
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      for (int k=0; k<n_target_neurons; k++) {
	for (int i=0; i<n_source_neurons; i++) {
	  int i_source_neuron = i + i_source_neuron_0;
	  
	  int i_remote_neuron = -1;
	  for (vector<ExternalConnectionNode >::iterator it =
		 connect_mpi_->extern_connection_[i_source_neuron].begin();
	       it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	    if ((*it).target_host_id == i_target_host) {
	      i_remote_neuron = (*it).remote_neuron_id;
	      break;
	    }
	  }
	  if (i_remote_neuron == -1) {
	    i_remote_neuron = i_new_remote_neuron;
	    i_new_remote_neuron++;
	    ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	    connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	  }
	  i_remote_neuron_arr[k*n_source_neurons+i] = i_remote_neuron;
	}
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_target_neurons*n_source_neurons,
				 i_target_host);
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnectOneToOne
(
 int i_source_host, int i_source_neuron_0,
 int i_target_host, int i_target_neuron_0, int n_neurons,
 unsigned char i_port, float weight, float delay
 )
{
  if (MpiId()==i_source_host && i_source_host==i_target_host) {
    return ConnectOneToOne(i_source_neuron_0, i_target_neuron_0,
			 n_neurons, i_port, weight, delay);
  }
  else if (MpiId()==i_source_host || MpiId()==i_target_host) {
    int *i_remote_neuron_arr = new int[n_neurons];
    int i_new_remote_neuron;
    if (MpiId() == i_target_host) {
      i_new_remote_neuron = net_connection_->connection_.size();
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_source_host);
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_source_host);
      vector<ConnGroup> conn;
      net_connection_->connection_.insert(net_connection_->connection_.end(),
					  i_new_remote_neuron
					  - net_connection_->connection_.size(), conn);
            
      //NEW, CHECK ///////////
      InsertNeuronGroup(i_new_remote_neuron
			- net_connection_->connection_.size(), 0);
      ///////////////////////
      
      connect_mpi_->MPI_Recv_int(i_remote_neuron_arr, n_neurons, i_source_host);

      for (int i=0; i<n_neurons; i++) {
	int i_remote_neuron = i_remote_neuron_arr[i];
	int i_target_neuron = i + i_target_neuron_0;
	net_connection_->Connect(i_remote_neuron, i_target_neuron, i_port, weight, delay);
      }
    }
    else if (MpiId() == i_source_host) {
      connect_mpi_->MPI_Recv_int(&i_new_remote_neuron, 1, i_target_host);
      for (int i=0; i<n_neurons; i++) {
	int i_source_neuron = i + i_source_neuron_0;
	  
	int i_remote_neuron = -1;
	for (vector<ExternalConnectionNode >::iterator it =
	       connect_mpi_->extern_connection_[i_source_neuron].begin();
	     it <  connect_mpi_->extern_connection_[i_source_neuron].end(); it++) {
	  if ((*it).target_host_id == i_target_host) {
	    i_remote_neuron = (*it).remote_neuron_id;
	    break;
	  }
	}
	if (i_remote_neuron == -1) {
	  i_remote_neuron = i_new_remote_neuron;
	  i_new_remote_neuron++;
	  ExternalConnectionNode conn_node = {i_target_host, i_remote_neuron};
	  connect_mpi_->extern_connection_[i_source_neuron].push_back(conn_node);
	}
	i_remote_neuron_arr[i] = i_remote_neuron;
      }
      connect_mpi_->MPI_Send_int(&i_new_remote_neuron, 1, i_target_host);
      connect_mpi_->MPI_Send_int(i_remote_neuron_arr, n_neurons, i_target_host);
    }
    delete[] i_remote_neuron_arr;
  }
  MPI_Barrier( MPI_COMM_WORLD );

  return 0;
}

int NeuralGPU::RemoteConnect(int i_source_host, int i_source_neuron,
			     int i_target_host, int i_target_neuron,
			     unsigned char i_port, float weight, float delay)
{
  return connect_mpi_->RemoteConnect(i_source_host, i_source_neuron,
				     i_target_host, i_target_neuron,
				     i_port, weight, delay);
}

int NeuralGPU::ConnectFixedIndegreeArray
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float *weight_arr, float *delay_arr, int indegree
 )
{
  unsigned int *rnd = RandomInt(n_target_neurons*indegree);
  vector<int> input_array;
  for (int i=0; i<n_source_neurons; i++) {
    input_array.push_back(i_source_neuron_0 + i);
  }
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared) collapse(2)
#endif
  for (int k=0; k<n_target_neurons; k++) {
    for (int i=0; i<indegree; i++) {
      int j = i + rnd[k*indegree+i] % (n_source_neurons - i);
#ifdef _OPENMP
      omp_set_lock(&(lock[i]));
#endif
      if (j!=i) {
#ifdef _OPENMP
	omp_set_lock(&(lock[j]));
#endif
	swap(input_array[i], input_array[j]);
#ifdef _OPENMP
	omp_unset_lock(&(lock[j]));
#endif
      }
      int itn = k + i_target_neuron_0;
      int isn = input_array[i];
      size_t i_arr = (size_t)k*indegree + i;
      net_connection_->Connect(isn, itn, i_port, weight_arr[i_arr],
			       delay_arr[i_arr]);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
    }
  }
  delete[] rnd;
#ifdef _OPENMP
  delete[] lock;
#endif
  
  return 0;
}

int NeuralGPU::ConnectFixedTotalNumberArray
(
 int i_source_neuron_0, int n_source_neurons,
 int i_target_neuron_0, int n_target_neurons,
 unsigned char i_port, float *weight_arr, float *delay_arr, int n_conn
 )
{
  unsigned int *rnd = RandomInt(2*n_conn);
#ifdef _OPENMP
  omp_lock_t *lock = new omp_lock_t[n_source_neurons];
  for (int i=0; i<n_source_neurons; i++) {
    omp_init_lock(&(lock[i]));
  }
#pragma omp parallel for default(shared)
#endif
  for (int i_conn=0; i_conn<n_conn; i_conn++) {
    int i = rnd[2*i_conn] % n_source_neurons;
    int j = rnd[2*i_conn+1] % n_target_neurons;
    int isn = i + i_source_neuron_0;
    int itn = j + i_target_neuron_0;
#ifdef _OPENMP
    omp_set_lock(&(lock[i]));
#endif
    net_connection_->Connect(isn, itn, i_port, weight_arr[i_conn],
                             delay_arr[i_conn]);
#ifdef _OPENMP
      omp_unset_lock(&(lock[i]));
#endif
  }
  delete[] rnd;
#ifdef _OPENMP
  delete[] lock;
#endif
  
  return 0;
}

unsigned int *NeuralGPU::RandomInt(size_t n)
{
  return curand_int(*random_generator_, n);
}

float *NeuralGPU::RandomUniform(size_t n)
{
  return hiprand_uniform(*random_generator_, n);
}

float *NeuralGPU::RandomNormal(size_t n, float mean, float stddev)
{
  return hiprand_normal(*random_generator_, n, mean, stddev);
}

float *NeuralGPU::RandomNormalClipped(size_t n, float mean, float stddev,
				      float vmin, float vmax)
{
  int n_extra = n/10;
  if (n_extra<1024) {
    n_extra=1024;
  }
  int i_extra = 0;
  float *arr = hiprand_normal(*random_generator_, n, mean, stddev);
  float *arr_extra;
  for (size_t i=0; i<n; i++) {
    while (arr[i]<vmin || arr[i]>vmax) {
      if (i_extra==0) {
	arr_extra = hiprand_normal(*random_generator_, n_extra, mean, stddev);
      }
      arr[i] = arr_extra[i_extra];
      i_extra++;
      if (i_extra==n_extra) {
	i_extra = 0;
	delete[](arr_extra);
      }
    }
  }
  if (i_extra != 0) {
    delete[](arr_extra);
  }
  
  return arr; 
}

