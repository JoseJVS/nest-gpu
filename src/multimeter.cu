/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>
#include <vector>
#include "multimeter.h"
#include "cuda_error.h"

using namespace std;

const   std::string SpikeVarName = "spike";

Record::Record(std::vector<BaseNeuron*> neur_vect, std::string file_name,
	       std::vector<std::string> var_name_vect,
	       std::vector<int> i_neur_vect, std::vector<int> i_port_vect):
  neuron_vect_(neur_vect), file_name_(file_name),
  var_name_vect_(var_name_vect),
  i_neuron_vect_(i_neur_vect),
  i_port_vect_(i_port_vect)
{
  data_vect_flag_ = true;
  if (file_name=="") {
    out_file_flag_ = false;
  } else {
    out_file_flag_ = true;
  }
  var_pt_vect_.clear();
  for (unsigned int i=0; i<var_name_vect.size(); i++) {
    if (var_name_vect[i]!=SpikeVarName) {
      float *var_pt = neur_vect[i]->GetVarPt(var_name_vect[i], i_neur_vect[i],
					     i_port_vect[i]);
      var_pt_vect_.push_back(var_pt);
    }
  }
}

int Record::OpenFile()
{
  fp_=fopen(file_name_.c_str(), "w");
	   
  return 0;
}

int Record::CloseFile()
{
  fclose(fp_);
	   
  return 0;
}

int Record::WriteRecord(float t)
{
  float var;
  vector<float> vect;
  
  if (out_file_flag_) {
    fprintf(fp_,"%f", t);
  }
  if (data_vect_flag_) {
    vect.push_back(t);
  }
  for (unsigned int i=0; i<var_name_vect_.size(); i++) {
    if (var_name_vect_[i]!=SpikeVarName) {
      gpuErrchk(hipMemcpy(&var, var_pt_vect_[i], sizeof(float),
			   hipMemcpyDeviceToHost));
    }
    else {
      var = neuron_vect_[i]->GetSpikeActivity(i_neuron_vect_[i]);
    }
    if (out_file_flag_) {
      fprintf(fp_,"\t%f", var);
    }
    if (data_vect_flag_) {
      vect.push_back(var);
    }
  }
  if (out_file_flag_) {
    fprintf(fp_,"\n");
  }
  if (data_vect_flag_) {
    data_vect_.push_back(vect);
  }

  return 0;
}

int Multimeter::CreateRecord(std::vector<BaseNeuron*> neur_vect,
			     std::string file_name,
			     std::vector<std::string> var_name_vect,
			     std::vector<int> i_neur_vect,
			     std::vector<int> i_port_vect)
{
  Record record(neur_vect, file_name, var_name_vect, i_neur_vect,
		i_port_vect);
  record_vect_.push_back(record);

  return (record_vect_.size() - 1);
}

int Multimeter::OpenFiles()
{
  for (unsigned int i=0; i<record_vect_.size(); i++) {
    if (record_vect_[i].out_file_flag_) {
      record_vect_[i].OpenFile();
    }
  }
  
  return 0;
}

int Multimeter::CloseFiles()
{  
  for (unsigned int i=0; i<record_vect_.size(); i++) {
    if (record_vect_[i].out_file_flag_) {
      record_vect_[i].CloseFile();
    }
  }
  
  return 0;
}

int Multimeter::WriteRecords(float t)
{  
  for (unsigned int i=0; i<record_vect_.size(); i++) {
    record_vect_[i].WriteRecord(t);
  }
  
  return 0;
}

std::vector<std::vector<float>> *Multimeter::GetRecordData(int i_record)
{
  if (i_record<0 || i_record>=(int)record_vect_.size()) {
    throw ngpu_exception("Record does not exist.");
  }
  
  return &record_vect_[i_record].data_vect_;
}
