#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>

#include <stdio.h>
#include <stdlib.h>

#include "cuda_error.h"
#include "spike_buffer.h"

#include "spike_mpi.h"
#include "connect_mpi.h"


__global__ void PushSpikeFromRemote(int n_spikes, int *spike_buffer_id,
           float *spike_height)
{
  int i_spike = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_spike<n_spikes) {
    int isb = spike_buffer_id[i_spike];
    float height = spike_height[i_spike];
    PushSpike(isb, height);
  }
}

__global__ void PushSpikeFromRemote(int n_spikes, int *spike_buffer_id)
{
  int i_spike = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_spike<n_spikes) {
    int isb = spike_buffer_id[i_spike];
    PushSpike(isb, 1.0);
  }
}

#ifdef HAVE_MPI

__constant__ bool NeuronGPUMpiFlag;

__device__ int NExternalTargetHost;
__device__ int MaxSpikePerHost;

int *d_ExternalSpikeNum;
__device__ int *ExternalSpikeNum;

int *d_ExternalSpikeSourceNode; // [MaxSpikeNum];
__device__ int *ExternalSpikeSourceNode;

float *d_ExternalSpikeHeight; // [MaxSpikeNum];
__device__ float *ExternalSpikeHeight;

int *d_ExternalTargetSpikeNum;
__device__ int *ExternalTargetSpikeNum;

int *d_ExternalTargetSpikeNodeId;
__device__ int *ExternalTargetSpikeNodeId;

float *d_ExternalTargetSpikeHeight;
__device__ float *ExternalTargetSpikeHeight;

int *d_NExternalNodeTargetHost;
__device__ int *NExternalNodeTargetHost;

int **d_ExternalNodeTargetHostId;
__device__ int **ExternalNodeTargetHostId;

int **d_ExternalNodeId;
__device__ int **ExternalNodeId;

//int *d_ExternalSourceSpikeNum;
//__device__ int *ExternalSourceSpikeNum;

int *d_ExternalSourceSpikeNodeId;
__device__ int *ExternalSourceSpikeNodeId;

float *d_ExternalSourceSpikeHeight;
__device__ float *ExternalSourceSpikeHeight;

int *h_ExternalSpikeNodeId;

float *h_ExternalSpikeHeight;

__device__ void PushExternalSpike(int i_source, float height)
{
  int pos = atomicAdd(ExternalSpikeNum, 1);
  ExternalSpikeSourceNode[pos] = i_source;
  ExternalSpikeHeight[pos] = height;
}

__global__ void SendExternalSpike()
{
  int i_spike = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_spike < *ExternalSpikeNum) {
    int i_source = ExternalSpikeSourceNode[i_spike];
    float height = ExternalSpikeHeight[i_spike];
    int Nth = NExternalNodeTargetHost[i_source];
      
    for (int ith=0; ith<Nth; ith++) {
      int target_host_id = ExternalNodeTargetHostId[i_source][ith];
      int remote_node_id = ExternalNodeId[i_source][ith];
      int pos = atomicAdd(&ExternalTargetSpikeNum[target_host_id], 1);
      ExternalTargetSpikeNodeId[target_host_id*MaxSpikePerHost + pos]
	= remote_node_id;
      ExternalTargetSpikeHeight[target_host_id*MaxSpikePerHost + pos]
	= height;
    }
  }
}

__global__ void ExternalSpikeReset()
{
  *ExternalSpikeNum = 0;
  for (int ith=0; ith<NExternalTargetHost; ith++) {
    ExternalTargetSpikeNum[ith] = 0;
  }
}

int ConnectMpi::ExternalSpikeInit(int n_node, int max_spike_num, int n_hosts,
				  int max_spike_per_host)
{
  int *h_NExternalNodeTargetHost = new int[n_node];
  int **h_ExternalNodeTargetHostId = new int*[n_node];
  int **h_ExternalNodeId = new int*[n_node];
  
  h_ExternalSpikeNodeId = new int[max_spike_num];

  h_ExternalSpikeHeight = new float[max_spike_num];
  
  gpuErrchk(hipMalloc(&d_ExternalSpikeNum, sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalSpikeSourceNode,
		       max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalSpikeHeight, max_spike_num*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalTargetSpikeNum, n_hosts*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalTargetSpikeNodeId,
		       n_hosts*max_spike_per_host*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalTargetSpikeHeight,
		       n_hosts*max_spike_per_host*sizeof(float)));
  //gpuErrchk(hipMalloc(&d_ExternalSourceSpikeNum, n_hosts*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalSourceSpikeNodeId, //n_hosts*
		       max_spike_per_host*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalSourceSpikeHeight, //n_hosts*
		       max_spike_per_host*sizeof(float)));
	    
  gpuErrchk(hipMalloc(&d_NExternalNodeTargetHost, n_node*sizeof(int)));
  gpuErrchk(hipMalloc(&d_ExternalNodeTargetHostId, n_node*sizeof(int*)));
  gpuErrchk(hipMalloc(&d_ExternalNodeId, n_node*sizeof(int*)));
 
  for (int i_source=0; i_source<n_node; i_source++) {
    std::vector< ExternalConnectionNode > *conn = &extern_connection_[i_source];
    int Nth = conn->size();
    h_NExternalNodeTargetHost[i_source] = Nth;
    if (Nth>0) {
       gpuErrchk(hipMalloc(&h_ExternalNodeTargetHostId[i_source],
   			 Nth*sizeof(int)));
       gpuErrchk(hipMalloc(&h_ExternalNodeId[i_source], Nth*sizeof(int)));
       int *target_host_arr = new int[Nth];
       int *node_id_arr = new int[Nth];
       for (int ith=0; ith<Nth; ith++) {
         target_host_arr[ith] = conn->at(ith).target_host_id;
         node_id_arr[ith] = conn->at(ith).remote_node_id;
       }
       hipMemcpy(h_ExternalNodeTargetHostId[i_source], target_host_arr,
   	       Nth*sizeof(int), hipMemcpyHostToDevice);
       hipMemcpy(h_ExternalNodeId[i_source], node_id_arr,
   	       Nth*sizeof(int), hipMemcpyHostToDevice);
       delete[] target_host_arr;
       delete[] node_id_arr;
     }
  }
  hipMemcpy(d_NExternalNodeTargetHost, h_NExternalNodeTargetHost,
	     n_node*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ExternalNodeTargetHostId, h_ExternalNodeTargetHostId,
	     n_node*sizeof(int*), hipMemcpyHostToDevice);
  hipMemcpy(d_ExternalNodeId, h_ExternalNodeId,
	     n_node*sizeof(int*), hipMemcpyHostToDevice);

  DeviceExternalSpikeInit<<<1,1>>>(n_hosts, max_spike_per_host,
				   d_ExternalSpikeNum,
				   d_ExternalSpikeSourceNode,
				   d_ExternalSpikeHeight,
				   d_ExternalTargetSpikeNum,
				   d_ExternalTargetSpikeNodeId,
				   d_ExternalTargetSpikeHeight,
				   d_NExternalNodeTargetHost,
				   d_ExternalNodeTargetHostId,
				   d_ExternalNodeId
				   );
  delete[] h_NExternalNodeTargetHost;
  delete[] h_ExternalNodeTargetHostId;
  delete[] h_ExternalNodeId;

  return 0;
}

__global__ void DeviceExternalSpikeInit(int n_hosts,
					int max_spike_per_host,
					int *ext_spike_num,
					int *ext_spike_source_node,
					float *ext_spike_height,
					int *ext_target_spike_num,
					int *ext_target_spike_node_id,
					float *ext_target_spike_height,
					int *n_ext_node_target_host,
					int **ext_node_target_host_id,
					int **ext_node_id
					)
  
{
  NExternalTargetHost = n_hosts;
  MaxSpikePerHost =  max_spike_per_host;
  ExternalSpikeNum = ext_spike_num;
  ExternalSpikeSourceNode = ext_spike_source_node;
  ExternalSpikeHeight = ext_spike_height;
  ExternalTargetSpikeNum = ext_target_spike_num;
  ExternalTargetSpikeNodeId = ext_target_spike_node_id;
  ExternalTargetSpikeHeight = ext_target_spike_height;
  NExternalNodeTargetHost = n_ext_node_target_host;
  ExternalNodeTargetHostId = ext_node_target_host_id;
  ExternalNodeId = ext_node_id;
  *ExternalSpikeNum = 0;
  for (int ith=0; ith<NExternalTargetHost; ith++) {
    ExternalTargetSpikeNum[ith] = 0;
  }  
}

int ConnectMpi::SendSpikeToRemote(int n_hosts, int max_spike_per_host)
{
  int mpi_id, tag = 1;  // id is already in the class, remove
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_id);

  int *h_ExternalTargetSpikeNum = new int[n_hosts];
  gpuErrchk(hipMemcpy(h_ExternalTargetSpikeNum, d_ExternalTargetSpikeNum,
                       n_hosts*sizeof(int), hipMemcpyDeviceToHost));
  for (int ih=0; ih<n_hosts; ih++) {
    if (ih == mpi_id) continue;
    int n_spike = h_ExternalTargetSpikeNum[ih];
    MPI_Send(&n_spike, 1, MPI_INT, ih, tag, MPI_COMM_WORLD);
    if (n_spike>0) {
      //cout << "nspike send: " << n_spike << endl;
#ifdef GPUDIRECT
      MPI_Send(&d_ExternalTargetSpikeNodeId[ih*max_spike_per_host],
	       n_spike, MPI_INT, ih, tag, MPI_COMM_WORLD);
      MPI_Send(&d_ExternalTargetSpikeHeight[ih*max_spike_per_host],
	       n_spike, MPI_FLOAT, ih, tag, MPI_COMM_WORLD);
#else
      gpuErrchk(hipMemcpy(h_ExternalSpikeNodeId,
			  &d_ExternalTargetSpikeNodeId[ih*max_spike_per_host],
			   n_spike*sizeof(int), hipMemcpyDeviceToHost));
      MPI_Send(h_ExternalSpikeNodeId,
               n_spike, MPI_INT, ih, tag, MPI_COMM_WORLD);
      gpuErrchk(hipMemcpy(h_ExternalSpikeHeight,
			  &d_ExternalTargetSpikeHeight[ih*max_spike_per_host],
			   n_spike*sizeof(float), hipMemcpyDeviceToHost));
      MPI_Send(h_ExternalSpikeHeight,
               n_spike, MPI_FLOAT, ih, tag, MPI_COMM_WORLD);
#endif      
    }
  }

  delete[] h_ExternalTargetSpikeNum;
  return 0;
}

int ConnectMpi::RecvSpikeFromRemote(int i_host, int max_spike_per_host)
{
  MPI_Status Stat;
  int mpi_id, tag = 1; // id is already in the class, remove
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_id);

  int n_spike;
  MPI_Recv(&n_spike, 1, MPI_INT, i_host, tag, MPI_COMM_WORLD, &Stat);
  //h_ExternalSourceSpikeNum[ih] = n_spike;
  if (n_spike>0) {
    //cout << "nspike recv: " << n_spike << endl;
#ifdef GPUDIRECT
    MPI_Recv(d_ExternalSourceSpikeNodeId, // [ih*max_spike_per_host],
	     n_spike, MPI_INT, i_host, tag, MPI_COMM_WORLD, &Stat);
    MPI_Recv(d_ExternalSourceSpikeHeight, // [ih*max_spike_per_host],
	     n_spike, MPI_FLOAT, i_host, tag, MPI_COMM_WORLD, &Stat);
#else
    MPI_Recv(h_ExternalSpikeNodeId,
	     n_spike, MPI_INT, i_host, tag, MPI_COMM_WORLD, &Stat);
    hipMemcpy(d_ExternalSourceSpikeNodeId, h_ExternalSpikeNodeId,
	       n_spike*sizeof(int), hipMemcpyHostToDevice);
    MPI_Recv(h_ExternalSpikeHeight,
	     n_spike, MPI_FLOAT, i_host, tag, MPI_COMM_WORLD, &Stat);
    hipMemcpy(d_ExternalSourceSpikeHeight, h_ExternalSpikeHeight,
	       n_spike*sizeof(float), hipMemcpyHostToDevice);
#endif
    PushSpikeFromRemote<<<(n_spike+1023)/1024, 1024>>>
      (n_spike, d_ExternalSourceSpikeNodeId,
      d_ExternalSourceSpikeHeight); //[ih*max_spike_per_host])
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    
  }

  //delete[] h_ExternalSourceSpikeNum;
  return 0;
}

#endif
