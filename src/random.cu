#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "cuda_error.h"

unsigned int *curand_int(hiprandGenerator_t &gen, size_t n)
{
  unsigned int *devData, *hostData;
  // Allocate n integers on host
  hostData = (unsigned int *)calloc(n, sizeof(unsigned int));
  // Allocate n integers on device
  CUDA_CALL(hipMalloc((void **)&devData, n * sizeof(unsigned int)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerate(gen, devData, n));
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(unsigned int),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(devData));
  
  return hostData;
}
