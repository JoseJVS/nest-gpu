#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "cuda_error.h"

unsigned int *curand_int(hiprandGenerator_t &gen, size_t n)
{
  unsigned int *dev_data;
  // Allocate n integers on host
  unsigned int *host_data = new unsigned int[n];
  
  // Allocate n integers on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(unsigned int)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerate(gen, dev_data, n));
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(unsigned int),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));
  
  return host_data;
}
