#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "cuda_error.h"

unsigned int *curand_int(hiprandGenerator_t &gen, size_t n)
{
  unsigned int *dev_data;
  // Allocate n integers on host
  unsigned int *host_data = new unsigned int[n];
  
  // Allocate n integers on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(unsigned int)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerate(gen, dev_data, n));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(unsigned int),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));
  
  return host_data;
}

float *hiprand_uniform(hiprandGenerator_t &gen, size_t n)
{
  float *dev_data;
  // Allocate n floats on host
  float *host_data = new float[n];
  
  // Allocate n floats on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(float)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerateUniform(gen, dev_data, n));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(float),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));
  
  return host_data;
}

float *hiprand_normal(hiprandGenerator_t &gen, size_t n, float mean,
		     float stddev)
{
  float *dev_data;
  // Allocate n floats on host
  float *host_data = new float[n];
  
  // Allocate n floats on device
  CUDA_CALL(hipMalloc((void **)&dev_data, n*sizeof(float)));
  // Create pseudo-random number generator

  // Generate n integers on device
  CURAND_CALL(hiprandGenerateNormal(gen, dev_data, n, mean, stddev));
  hipDeviceSynchronize();
  // Copy device memory to host
  CUDA_CALL(hipMemcpy(host_data, dev_data, n*sizeof(float),
                       hipMemcpyDeviceToHost));
  // Cleanup
  CUDA_CALL(hipFree(dev_data));
  
  return host_data;
}

